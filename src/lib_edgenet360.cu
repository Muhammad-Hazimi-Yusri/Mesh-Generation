/*
    EdgeNet data preprocessing adaped to 360 degrees images
    Adapted to use with Python and numpy
    Author: Aloísio Dourado (jun, 2018)
    Original Caffe Code: Shuran Song (https://github.com/shurans/sscnet)
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

typedef high_resolution_clock::time_point clock_tick;
#define MIN(X, Y) (((X) <= (Y)) ? (X) : (Y))
#define MAX(X, Y) (((X) >= (Y)) ? (X) : (Y))

// Voxel information
float vox_unit = 0.02;
float vox_margin = 0.24;
int vox_size_x = 240;
int vox_size_y = 144;
int vox_size_z = 240;

// Camera information
float f = 518.85;
float sensor_w = 640;
float sensor_h = 480;

// GPU parameters
int NUM_THREADS=1024;
int DEVICE = 0;
int debug = 0;

// GPU Variables
float *parameters_GPU;
#define VOX_UNIT (0)
#define VOX_MARGIN (1)
#define VOX_SIZE_X (2)
#define VOX_SIZE_Y (3)
#define VOX_SIZE_Z (4)
#define CAM_F (5)
#define SENSOR_W (6)
#define SENSOR_H (7)

#define GO_BACK (1.7) //Backward camera for better FOV (meters)
#define CAM_HEIGHT (1.0) //reposition camera for better FOV (meters)

#define FLOOR_OFFSET (0.00)
//VOX_LIMITS
#define OUT_OF_FOV (4)
#define OUT_OF_ROOM (3)
#define OCCLUDED (2)
#define OCCUPIED (1)
#define EMPTY_VISIBLE (0)



#define NUM_CLASSES (256)
#define MAX_DOWN_SIZE (1000)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

clock_tick start_timer(){
    return (high_resolution_clock::now());
}

void end_timer(clock_tick t1, const char msg[]) {
  if (debug==1){
      clock_tick t2 = high_resolution_clock::now();
      auto duration = duration_cast<milliseconds>( t2 - t1 ).count();
      printf("%s: %ld(ms)\n", msg, duration);
  }
}

//float cam_K[9] = {518.8579f, 0.0f, (float)frame_width / 2.0f, 0.0f, 518.8579f, (float)frame_height / 2.0f, 0.0f, 0.0f, 1.0f};




void setup_CPP(int device, int num_threads, float v_unit, float v_margin,
               float focal_length, float s_w, float s_h,
               int vox_x, int vox_y, int vox_z,
               int debug_flag){
    DEVICE = device;
    NUM_THREADS = num_threads;
    vox_unit = v_unit;
    vox_margin = v_margin;
    f = focal_length;
    sensor_w = s_w;
    sensor_h = s_h;
    vox_size_x = vox_x;
    vox_size_y = vox_y;
    vox_size_z = vox_z;

    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, DEVICE);
    hipSetDevice(DEVICE);

    if (debug_flag==1) {

        printf("\nUsing GPU: %s - (device %d)\n", deviceProperties.name, DEVICE);
        printf("Total Memory: %ld\n", deviceProperties.totalGlobalMem);
        printf("Max threads per block: %d\n", deviceProperties.maxThreadsPerBlock);
        printf("Max threads dimension: (%d, %d, %d)\n", deviceProperties.maxGridSize[0],
                                                        deviceProperties.maxGridSize[1],
                                                        deviceProperties.maxGridSize[2]);
        printf("Major, Minor: (%d, %d)\n", deviceProperties.major, deviceProperties.minor);
        printf("Multiprocessor count: %d\n", deviceProperties.multiProcessorCount);
        printf("Threads per block: %d\n", NUM_THREADS);
    }

    debug = debug_flag;

    if (NUM_THREADS>deviceProperties.maxThreadsPerBlock){
        printf("Selected NUM_THREADS (%d) is greater than device's max threads per block (%d)\n",
               NUM_THREADS, deviceProperties.maxThreadsPerBlock);
        exit(0);
    }


    float parameters[8];

    hipMalloc(&parameters_GPU, 8 * sizeof(float));

    parameters[VOX_UNIT] = vox_unit;
    parameters[VOX_MARGIN] = vox_margin;
    parameters[CAM_F] = f;
    parameters[SENSOR_W] = sensor_w;
    parameters[SENSOR_H] = sensor_h;
    parameters[VOX_SIZE_X] = (float)vox_size_x;
    parameters[VOX_SIZE_Y] = (float)vox_size_y;
    parameters[VOX_SIZE_Z] = (float)vox_size_z;


    hipMemcpy(parameters_GPU, parameters, 8 * sizeof(float), hipMemcpyHostToDevice);


}

void clear_parameters_GPU(){
    hipFree(parameters_GPU);
}


__global__
void point_cloud_kernel(float *baseline, unsigned char *depth_data,
                        float *point_cloud, int *width, int *height){

  //if (threadIdx.x==0) printf("fwg %d  fwg %d", frame_width_GPU,frame_height_GPU);

  //Rerieve pixel coodinates
  int pixel_idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (pixel_idx >= (*width * *height))
    return;

  int pixel_y = pixel_idx / *width;
  int pixel_x = pixel_idx % *width;

  //if (threadIdx.x==0 ) {printf("blockIdx.x:%d pidx:%d px:%d py:%d\n", blockIdx.x, pixel_idx, pixel_x, pixel_y );}

  float     CV_PI = 3.141592;

  int		max_radius = 30;
  int		inf_border = 160;		// Range (in pixel) from the pole to exclude from point cloud generation
  double	unit_h, unit_w;	//angular size of 1 pixel
  float		disp_scale = 2;
  float		disp_offset = -120;

  unit_h = 1.0 / (*height);
  unit_w = 2.0 / (*width);

  // Get point in world coordinate
  // Try to parallel later

  int point_disparity = depth_data[pixel_y * *width + pixel_x];


  float longitude, latitude, radius, angle_disp;

  latitude = pixel_y * unit_h * CV_PI;

  longitude = pixel_x * unit_w * CV_PI;

  point_cloud[6 * pixel_idx + 3] = latitude;
  point_cloud[6 * pixel_idx + 4] = longitude;

  if (point_disparity == 0)
	return;

  if (pixel_y<inf_border || pixel_y> *height - inf_border)
	return;

  angle_disp = (point_disparity / disp_scale + disp_offset) * unit_h * CV_PI;

  if (latitude + angle_disp <0)
    angle_disp = 0.01;

  if (angle_disp == 0)   {
	radius = max_radius;
	point_disparity = 0;
  }	else
	radius = *baseline / ((sin(latitude) / tan(latitude + angle_disp)) - cos(latitude));

  if (radius > max_radius || radius < 0.0) 	{
	radius = max_radius;
	point_disparity = 0;
  }

  //too close
  //if (latitude < CV_PI/4) || (latitude > CV_PI - CV_PI/4))
  //if (latitude < CV_PI/3)
  //  return;


  //world coordinates
  //float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  //float ry = radius*sin(latitude)*sin(CV_PI - longitude);
  //float rz = radius*cos(latitude);
  //voxel coordinates
  //int z = (int)floor(rz / vox_unit_GPU + vox_size[2]/2);
  //int x = (int)floor(rx / vox_unit_GPU + vox_size[0]/2));
  //int y = (int)floor(ry / vox_unit_GPU);


  //float rx = -radius*sin(latitude)*cos(CV_PI - longitude);
  float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  float rz = radius*sin(latitude)*sin(CV_PI - longitude);
  float ry = radius*cos(latitude); //+.20cm to get the floor

  //voxel coordinates
  //int z = (int)floor(rz / vox_unit_GPU);
  //int x = (int)floor(rx / vox_unit_GPU);// + vox_size[0]/2);
  //int y = (int)floor(ry / vox_unit_GPU);// + vox_size[1]/2);

  point_cloud[6 * pixel_idx + 0] = rx;
  point_cloud[6 * pixel_idx + 1] = ry;
  point_cloud[6 * pixel_idx + 2] = rz;
  point_cloud[6 * pixel_idx + 5] = radius;
  //if (threadIdx.x==0 ) {printf("blockIdx.x:%d pcx:%2.2f rx:%2.2f ry:%2.2f rz:%2.2f lat:%3.0f long:%3.0f \n",
  //                      blockIdx.x, point_cloud[6 * pixel_idx + 0], rx, ry, rz, latitude*180/CV_PI, longitude*180/CV_PI);}

}


__global__
void point_cloud_stanford_kernel(uint16_t *depth_data,
                        float *point_cloud, int *width, int *height){

  //if (threadIdx.x==0) printf("fwg %d  fwg %d", frame_width_GPU,frame_height_GPU);

  //Rerieve pixel coodinates
  int pixel_idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (pixel_idx >= (*width * *height))
    return;

  int pixel_y = pixel_idx / *width;
  int pixel_x = pixel_idx % *width;

  //if (threadIdx.x==0 ) {printf("blockIdx.x:%d pidx:%d px:%d py:%d\n", blockIdx.x, pixel_idx, pixel_x, pixel_y );}

  float     CV_PI = 3.141592;

  int		max_radius = 30;
  //int		inf_border = 160;		// Range (in pixel) from the pole to exclude from point cloud generation
  double	unit_h, unit_w;	//angular size of 1 pixel

  unit_h = 1.0 / (*height);
  unit_w = 2.0 / (*width);

  // Get point in world coordinate
  // Try to parallel later

  uint16_t point_depth = depth_data[pixel_y * *width + pixel_x];


  float longitude, latitude, radius;

  latitude = pixel_y * unit_h * CV_PI;

  longitude = pixel_x * unit_w * CV_PI;

  point_cloud[6 * pixel_idx + 3] = latitude;
  point_cloud[6 * pixel_idx + 4] = longitude;

  if (point_depth == 65535)
	return;

  //if (pixel_y<inf_border || pixel_y> *height - inf_border)
  //	return;

  radius = point_depth / 512.0;

  if (radius > max_radius || radius < 0.0) 	{
	radius = max_radius;
	point_depth = 0;
  }

  //too close
  //if (latitude < CV_PI/4) || (latitude > CV_PI - CV_PI/4))
  //if (latitude < CV_PI/3)
  //  return;


  //world coordinates
  //float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  //float ry = radius*sin(latitude)*sin(CV_PI - longitude);
  //float rz = radius*cos(latitude);
  //voxel coordinates
  //int z = (int)floor(rz / vox_unit_GPU + vox_size[2]/2);
  //int x = (int)floor(rx / vox_unit_GPU + vox_size[0]/2));
  //int y = (int)floor(ry / vox_unit_GPU);


  //float rx = -radius*sin(latitude)*cos(CV_PI - longitude);
  float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  float rz = radius*sin(latitude)*sin(CV_PI - longitude);
  float ry = radius*cos(latitude); //+.20cm to get the floor

  //voxel coordinates
  //int z = (int)floor(rz / vox_unit_GPU);
  //int x = (int)floor(rx / vox_unit_GPU);// + vox_size[0]/2);
  //int y = (int)floor(ry / vox_unit_GPU);// + vox_size[1]/2);

  point_cloud[6 * pixel_idx + 0] = rx;
  point_cloud[6 * pixel_idx + 1] = ry;
  point_cloud[6 * pixel_idx + 2] = rz;
  point_cloud[6 * pixel_idx + 5] = radius;
  //if (threadIdx.x==0 ) {printf("blockIdx.x:%d pcx:%2.2f rx:%2.2f ry:%2.2f rz:%2.2f lat:%3.0f long:%3.0f \n",
  //                      blockIdx.x, point_cloud[6 * pixel_idx + 0], rx, ry, rz, latitude*180/CV_PI, longitude*180/CV_PI);}

}

void get_point_cloud_CPP(float baseline, unsigned char *depth_data, float *point_cloud, int width, int height) {

  clock_tick t1 = start_timer();

  float *baseline_GPU;
  int *width_GPU;
  int *height_GPU;
  unsigned char *depth_data_GPU;
  float *point_cloud_GPU;

  int num_pixels = width * height;


  gpuErrchk(hipMalloc(&baseline_GPU, sizeof(float)));
  gpuErrchk(hipMalloc(&width_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&height_GPU, sizeof(int)));

  gpuErrchk(hipMalloc(&depth_data_GPU, num_pixels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&point_cloud_GPU, 6 * num_pixels * sizeof(float)));
  gpuErrchk(hipMemset(point_cloud_GPU, 0, 6 * num_pixels * sizeof(float)));

  gpuErrchk(hipMemcpy(baseline_GPU, &baseline, sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(width_GPU, &width, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(height_GPU, &height, sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(depth_data_GPU, depth_data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice));

  end_timer(t1, "Prepare duration");

  if (debug==1) printf("frame width: %d   frame heigth: %d   num_pixels %d\n" , width,height, num_pixels);


  t1 = start_timer();
  // from depth map to binaray voxel representation
  //depth2Grid<<<frame_width,frame_height>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);


  int NUM_BLOCKS = int((width*height + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  point_cloud_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(baseline_GPU, depth_data_GPU, point_cloud_GPU,
                                                  width_GPU, height_GPU);

  gpuErrchk( hipPeekAtLastError() );

  gpuErrchk( hipDeviceSynchronize() );

  end_timer(t1,"depth2Grid duration");

  hipMemcpy(point_cloud, point_cloud_GPU,  6* num_pixels * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(baseline_GPU);
  hipFree(width_GPU);
  hipFree(height_GPU);
  hipFree(depth_data_GPU);
  hipFree(point_cloud_GPU);

  end_timer(t1,"closeup duration");

}

void get_point_cloud_stanford_CPP(uint16_t *depth_data, float *point_cloud, int width, int height) {

  clock_tick t1 = start_timer();

  int *width_GPU;
  int *height_GPU;
  uint16_t *depth_data_GPU;
  float *point_cloud_GPU;

  int num_pixels = width * height;


  gpuErrchk(hipMalloc(&width_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&height_GPU, sizeof(int)));

  gpuErrchk(hipMalloc(&depth_data_GPU, num_pixels * sizeof(uint16_t)));
  gpuErrchk(hipMalloc(&point_cloud_GPU, 6 * num_pixels * sizeof(float)));
  gpuErrchk(hipMemset(point_cloud_GPU, 0, 6 * num_pixels * sizeof(float)));

  gpuErrchk(hipMemcpy(width_GPU, &width, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(height_GPU, &height, sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(depth_data_GPU, depth_data, height * width * sizeof(uint16_t), hipMemcpyHostToDevice));

  end_timer(t1, "Prepare duration");

  if (debug==1) printf("frame width: %d   frame heigth: %d   num_pixels %d\n" , width,height, num_pixels);


  t1 = start_timer();
  // from depth map to binaray voxel representation
  //depth2Grid<<<frame_width,frame_height>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);


  int NUM_BLOCKS = int((width*height + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  point_cloud_stanford_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(depth_data_GPU, point_cloud_GPU,
                                                  width_GPU, height_GPU);

  gpuErrchk( hipPeekAtLastError() );

  gpuErrchk( hipDeviceSynchronize() );

  end_timer(t1,"depth2Grid duration");

  hipMemcpy(point_cloud, point_cloud_GPU,  6* num_pixels * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(width_GPU);
  hipFree(height_GPU);
  hipFree(depth_data_GPU);
  hipFree(point_cloud_GPU);

  end_timer(t1,"closeup duration");

}

__global__
void get_voxels_kernel(float *point_cloud_GPU, int *point_cloud_size_GPU,
                       float *boundaries_GPU, int *vol_number_GPU, unsigned char *vox_grid_GPU, float *parameters_GPU){

  //if (blockIdx.x!=2000)
  //   return;
  //printf("boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
  //                     boundaries_GPU[0], boundaries_GPU[1], boundaries_GPU[2], boundaries_GPU[3], boundaries_GPU[4], boundaries_GPU[5]);

  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x: %d blockIdx.x:%d point_cloud_size:%d P0!!\n", threadIdx.x, blockIdx.x, *point_cloud_size_GPU);}

  //Rerieve pixel coodinates
  int point_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (point_idx >= *point_cloud_size_GPU)
    return;

  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x: %d blockIdx.x:%d  point_idx:%d P1!!\n", threadIdx.x, blockIdx.x, point_idx);}

  int x_idx = point_idx * 6 + 0;
  int y_idx = point_idx * 6 + 1;
  int z_idx = point_idx * 6 + 2;
  int lat_idx = point_idx * 6 + 3;
  int long_idx = point_idx * 6 + 4;
  int rd_idx = point_idx * 6 + 5;

  float  min_x = boundaries_GPU[0];
  float  max_x = boundaries_GPU[1];
  float  min_y = boundaries_GPU[2];
  float  max_y = boundaries_GPU[3];
  float  min_z = boundaries_GPU[4];
  float  max_z = boundaries_GPU[5];

  //int GO_DOWN = -min_y -EMUL_CAMERA_HEIGHT;

  float wx = point_cloud_GPU[x_idx];
  float wy = point_cloud_GPU[y_idx];
  float wz = point_cloud_GPU[z_idx];
  float latitude = point_cloud_GPU[lat_idx];
  float longitude = point_cloud_GPU[long_idx];
  float rd = point_cloud_GPU[rd_idx];

  float vox_unit_GPU = parameters_GPU[VOX_UNIT];
  float sensor_w_GPU = parameters_GPU[SENSOR_W];
  float sensor_h_GPU = parameters_GPU[SENSOR_H];
  float f_GPU = parameters_GPU[CAM_F];
  int vox_size_x_GPU = (int)parameters_GPU[VOX_SIZE_X];
  int vox_size_y_GPU = (int)parameters_GPU[VOX_SIZE_Y];
  int vox_size_z_GPU = (int)parameters_GPU[VOX_SIZE_Z];

  if ((wx == 0.) && (wy == 0.) && (wz == 0.)) {
    //if (blockIdx.x >40 && blockIdx.x <45) {printf("ZERO idx:%d rx:%f ry:%f rz:%f\n",point_idx, wx, wy, wz);}
    return;
  }


  if ((wx < min_x) || (wx > max_x) || (wy < min_y) || (wy > max_y) || (wz < min_z) || (wz > max_z) ) {
    //printf("OUT OF BOUNDARIES idx:%d rx: %2.2f (%2.2f %2.2f) ry: %2.2f (%2.2f %2.2f) rz: %2.2f (%2.2f %2.2f)\n",
    //        point_idx, wx, min_x, max_x, wy, min_y, max_y, wz, min_z, max_z);
    return;
  }

  /**/
  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x:%d blockIdx.x:%d %2.2f %2.2f %2.2f P2!!\n", threadIdx.x, blockIdx.x, wx, wy, wz);}

  int vx, vy, vz;
  //Adjust to vol_number
  if (*vol_number_GPU == 1) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(wx/vox_unit_GPU + vox_size_z_GPU/2);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 2) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (wx + wz) * (wx + wz))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 3) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU + vox_size_z_GPU/2);
      vx = (int)floor(wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 4) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wx/vox_unit_GPU);
      vx = (int)floor(-wz/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 5) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU + vox_size_x_GPU/2);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }
  if (*vol_number_GPU == 6) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 7) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU + vox_size_x_GPU/2);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }


  if (*vol_number_GPU == 8) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }



  // mark vox_out with 1.0
  if( vx >= 0 && vx < vox_size_x_GPU && vy >= 0 && vy < vox_size_y_GPU && vz >= 0 && vz < vox_size_z_GPU){
      int vox_idx = vz * vox_size_x_GPU * vox_size_y_GPU + vy * vox_size_x_GPU + vx;
      vox_grid_GPU[vox_idx] = float(1.0);


      //Edges


  } else {
      //printf("OUT idx:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n", point_idx, wx, wy, wz, vx, vy, vz);
  }


}

__global__
void get_voxels_edge_kernel(float *point_cloud_GPU, int *width_GPU, int *height_GPU,
                       float *boundaries_GPU, int *vol_number_GPU, unsigned char *vox_grid_edges_GPU,
                       unsigned char *edges_data_GPU, float *parameters_GPU){

  //if (blockIdx.x!=2000)
  //   return;
  //printf("boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
  //                     boundaries_GPU[0], boundaries_GPU[1], boundaries_GPU[2], boundaries_GPU[3], boundaries_GPU[4], boundaries_GPU[5]);

  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x: %d blockIdx.x:%d point_cloud_size:%d P0!!\n", threadIdx.x, blockIdx.x, *point_cloud_size_GPU);}

  //Rerieve pixel coodinates

  int point_cloud_size = *width_GPU * *height_GPU;

  int point_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (point_idx >= point_cloud_size)
    return;

  int pixel_y = point_idx / *width_GPU;
  int pixel_x = point_idx % *width_GPU;


  unsigned char point_edges = edges_data_GPU[pixel_y * *width_GPU + pixel_x];

   if (point_edges == 0) {
     return;
   }

  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x: %d blockIdx.x:%d  point_idx:%d P1!!\n", threadIdx.x, blockIdx.x, point_idx);}



  float min_depth = point_cloud_GPU[point_idx + 5];
  int closest_x = pixel_x;
  int closest_y = pixel_y;

  //Search for the closest depth around the edge to get the object at the foreground
  for (int x =  pixel_x - 1; x<=pixel_x+1; x++) {
      if (x>=0 & x<*width_GPU) {
          for (int y = pixel_y -1; y<=pixel_y+1; y++) {
               if (y>=0 & y<*height_GPU) {

                      point_idx = y * *width_GPU + x;

                      float point_depth = point_cloud_GPU[point_idx + 5];
                      if (point_depth < min_depth) {
                               min_depth = point_depth;
                               closest_x = x;
                               closest_y = y;
                      }
               }
          }
      }

  }

  point_idx = closest_y * *width_GPU + closest_x;

  int x_idx = point_idx * 6 + 0;
  int y_idx = point_idx * 6 + 1;
  int z_idx = point_idx * 6 + 2;
  int lat_idx = point_idx * 6 + 3;
  int long_idx = point_idx * 6 + 4;
  int rd_idx = point_idx * 6 + 5;

  float  min_x = boundaries_GPU[0];
  float  max_x = boundaries_GPU[1];
  float  min_y = boundaries_GPU[2];
  float  max_y = boundaries_GPU[3];
  float  min_z = boundaries_GPU[4];
  float  max_z = boundaries_GPU[5];

  float wx = point_cloud_GPU[x_idx];
  float wy = point_cloud_GPU[y_idx];
  float wz = point_cloud_GPU[z_idx];
  float latitude = point_cloud_GPU[lat_idx];
  float longitude = point_cloud_GPU[long_idx];
  float rd = point_cloud_GPU[rd_idx];

  float vox_unit_GPU = parameters_GPU[VOX_UNIT];
  float sensor_w_GPU = parameters_GPU[SENSOR_W];
  float sensor_h_GPU = parameters_GPU[SENSOR_H];
  float f_GPU = parameters_GPU[CAM_F];
  int vox_size_x_GPU = (int)parameters_GPU[VOX_SIZE_X];
  int vox_size_y_GPU = (int)parameters_GPU[VOX_SIZE_Y];
  int vox_size_z_GPU = (int)parameters_GPU[VOX_SIZE_Z];

  if ((wx == 0.) && (wy == 0.) && (wz == 0.)) {
    //if (blockIdx.x >40 && blockIdx.x <45) {printf("ZERO idx:%d rx:%f ry:%f rz:%f\n",point_idx, wx, wy, wz);}
    return;
  }


  if ((wx < min_x) || (wx > max_x) || (wy < min_y) || (wy > max_y) || (wz < min_z) || (wz > max_z) ) {
    //printf("OUT OF BOUNDARIES idx:%d rx: %2.2f (%2.2f %2.2f) ry: %2.2f (%2.2f %2.2f) rz: %2.2f (%2.2f %2.2f)\n",
    //        point_idx, wx, min_x, max_x, wy, min_y, max_y, wz, min_z, max_z);
    return;
  }

  /**/
  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x:%d blockIdx.x:%d %2.2f %2.2f %2.2f P2!!\n", threadIdx.x, blockIdx.x, wx, wy, wz);}

  int vx, vy, vz;
  //Adjust to vol_number
  if (*vol_number_GPU == 1) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(wx/vox_unit_GPU + vox_size_z_GPU/2);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 2) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (wx + wz) * (wx + wz))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 3) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU + vox_size_z_GPU/2);
      vx = (int)floor(wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 4) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wx/vox_unit_GPU);
      vx = (int)floor(-wz/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 5) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU + vox_size_x_GPU/2);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }
  if (*vol_number_GPU == 6) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }

  if (*vol_number_GPU == 7) {  //Vol 1 has no adjustments

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(-wz/vox_unit_GPU + vox_size_x_GPU/2);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }


  if (*vol_number_GPU == 8) {  //Vol 1 has no adjustments

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        //printf("OUT OF FOV(%2.2f %2.2f %2.2f %2.2f %2.2f %2.2f) idx:%d rx:%2.2f ry:%2.2f rz:%2.2f\n",
        //sensor_w_GPU, sensor_h_GPU, f_GPU, rd, fov_x, fov_y,point_idx, wx, wy, wz);
        return;
      }

      //voxel coordinates
      vz = (int)floor(wz/vox_unit_GPU);
      vx = (int)floor(-wx/vox_unit_GPU);
      //int vy = (int)floor(wy/vox_unit_GPU + vox_size_y_GPU/2);
      vy = (int)floor((wy - (min_y-FLOOR_OFFSET))/vox_unit_GPU);

  }



  // mark vox_out with 1.0
  if( vx >= 0 && vx < vox_size_x_GPU && vy >= 0 && vy < vox_size_y_GPU && vz >= 0 && vz < vox_size_z_GPU){
      int vox_idx = vz * vox_size_x_GPU * vox_size_y_GPU + vy * vox_size_x_GPU + vx;
      vox_grid_edges_GPU[vox_idx] = (unsigned char)1;

  } else {
      //printf("OUT idx:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n", point_idx, wx, wy, wz, vx, vy, vz);
  }


}


__global__
void filter_voxels_kernel(unsigned char *vox_grid_in_GPU, unsigned char *vox_grid_out_GPU, float *parameters_GPU){

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
    float downscale = 4;

    int vox_size_x = (int)parameters_GPU[VOX_SIZE_X];
    int vox_size_y = (int)parameters_GPU[VOX_SIZE_Y];
    int vox_size_z = (int)parameters_GPU[VOX_SIZE_Z];

    if (vox_idx >= vox_size_x * vox_size_y * vox_size_z){
      return;
    }

    if (vox_grid_in_GPU[vox_idx] == 0) {
        vox_grid_out_GPU[vox_idx] = 0;
        return;
    }

    int z = (vox_idx / ( vox_size_x * vox_size_y))%vox_size_z ;
    int y = (vox_idx / vox_size_x) % vox_size_y;
    int x = vox_idx % vox_size_x;

    int sum_occupied = 0;

    for (int tmp_x = MAX(0,x-downscale/2); tmp_x < MIN(x+downscale/2,vox_size_x); ++tmp_x) {
      for (int tmp_y = MAX(0,y-downscale/2); tmp_y < MIN(y+downscale/2,vox_size_y); ++tmp_y) {
        for (int tmp_z = MAX(0,z-downscale/2); tmp_z < MIN(z+downscale/2,vox_size_z); ++tmp_z) {

          int tmp_vox_idx = tmp_z * vox_size_x * vox_size_y + tmp_y * vox_size_z + tmp_x;

          if (vox_grid_in_GPU[tmp_vox_idx]> 0){
            sum_occupied += 1;          }
        }
      }
    }
    if (sum_occupied>=8) {  //empty threshold
      vox_grid_out_GPU[vox_idx] = 1;
    } else {
      vox_grid_out_GPU[vox_idx] = 0;
    }

}




void get_voxels_CPP(float *point_cloud, unsigned char *edges_data, int width, int height, float *boundaries, int vol_number,
                    unsigned char *vox_grid, unsigned char *vox_grid_edges) {

  clock_tick t1 = start_timer();

  int point_cloud_size = width * height;

  float *point_cloud_GPU;
  unsigned char *edges_data_GPU;
  int *point_cloud_size_GPU;
  float *boundaries_GPU;
  int *vol_number_GPU;
  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_edges_GPU;
  int *width_GPU, *height_GPU;
  //unsigned char *filtered_vox_grid_GPU;

  int num_voxels = vox_size_x * vox_size_y * vox_size_z;

  if (debug==1) printf("get_voxels - point_cloud_size: %d   vol_number: %d  voxel_size: %d %d %d\n" ,
                       point_cloud_size, vol_number, vox_size_x , vox_size_y , vox_size_z);

  if (debug==1) printf("get_voxels - boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
                       boundaries[0], boundaries[1], boundaries[2], boundaries[3], boundaries[4], boundaries[5]);

  gpuErrchk(hipMalloc(&edges_data_GPU, point_cloud_size * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&point_cloud_GPU, point_cloud_size * 6 * sizeof(float)));
  gpuErrchk(hipMalloc(&point_cloud_size_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&boundaries_GPU, 6 * sizeof(float)));
  gpuErrchk(hipMalloc(&vol_number_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&width_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&height_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&vox_grid_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_edges_GPU, num_voxels * sizeof(unsigned char)));
  //gpuErrchk(cudaMalloc(&filtered_vox_grid_GPU, num_voxels * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(point_cloud_GPU, point_cloud, point_cloud_size * 6 * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(edges_data_GPU, edges_data, point_cloud_size * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(point_cloud_size_GPU, &point_cloud_size, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(boundaries_GPU, boundaries, 6 * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vol_number_GPU, &vol_number, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(width_GPU, &width, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(height_GPU, &height, sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(vox_grid_GPU, 0, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMemset(vox_grid_edges_GPU, 0, num_voxels * sizeof(unsigned char)));

  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((point_cloud_size + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("get_voxels - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  get_voxels_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(point_cloud_GPU, point_cloud_size_GPU,
                                                 boundaries_GPU, vol_number_GPU, vox_grid_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  get_voxels_edge_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(point_cloud_GPU, width_GPU, height_GPU,
                                                 boundaries_GPU, vol_number_GPU, vox_grid_edges_GPU, edges_data_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  end_timer(t1,"get_voxels duration");

  //cudaMemcpy(vox_grid, filtered_vox_grid_GPU,  num_voxels * sizeof(unsigned char), cudaMemcpyDeviceToHost);
  hipMemcpy(vox_grid, vox_grid_GPU,  num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);
  hipMemcpy(vox_grid_edges, vox_grid_edges_GPU,  num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(point_cloud_GPU);
  hipFree(point_cloud_size_GPU);
  hipFree(boundaries_GPU);
  hipFree(vol_number_GPU);
  hipFree(width_GPU);
  hipFree(height_GPU);
  hipFree(vox_grid_GPU);
  hipFree(vox_grid_edges_GPU);
  hipFree(edges_data_GPU);

  end_timer(t1,"cleanup duration");

}

__global__
void get_one_hot_kernel(float *point_cloud_GPU, int *point_cloud_size_GPU,
                       float *boundaries_GPU, int *one_hot_GPU, float *parameters_GPU){

  //if (blockIdx.x!=2000)
  //   return;
  //printf("boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
  //                     boundaries_GPU[0], boundaries_GPU[1], boundaries_GPU[2], boundaries_GPU[3], boundaries_GPU[4], boundaries_GPU[5]);

  //if (blockIdx.x >40 && blockIdx.x <45) {printf("threadIdx.x: %d blockIdx.x:%d point_cloud_size:%d P0!!\n", threadIdx.x, blockIdx.x, *point_cloud_size_GPU);}

  //Rerieve pixel coodinates
  int point_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (point_idx >= *point_cloud_size_GPU)
    return;



  float wx = point_cloud_GPU[point_idx * 4 + 0];
  float wy = point_cloud_GPU[point_idx * 4 + 1];
  float wz = point_cloud_GPU[point_idx * 4 + 2];
  float lbl = point_cloud_GPU[point_idx * 4 + 3];

  float  min_x = boundaries_GPU[0];
  float  max_x = boundaries_GPU[1];
  float  min_y = boundaries_GPU[2];
  float  max_y = boundaries_GPU[3];
  float  min_z = boundaries_GPU[4];
  float  max_z = boundaries_GPU[5];

  if ((wx < min_x) || (wx > max_x) || (wy < min_y) || (wy > max_y) || (wz < min_z) || (wz > max_z) ) {
  //  printf("OUT OF BOUNDARIES idx:%d rx: %2.2f (%2.2f %2.2f) ry: %2.2f (%2.2f %2.2f) rz: %2.2f (%2.2f %2.2f)\n",
  //          point_idx, wx, min_x, max_x, wy, min_y, max_y, wz, min_z, max_z);
    return;
  }


  float vox_unit_GPU = parameters_GPU[VOX_UNIT]*4; //I am downsampling here

  int vox_size_x_GPU = (int)(parameters_GPU[VOX_SIZE_X]/2);
  int vox_size_y_GPU = (int)(parameters_GPU[VOX_SIZE_Y]/4);
  int vox_size_z_GPU = (int)(parameters_GPU[VOX_SIZE_Z]/2);

  int vx =  (int) floor(wx/vox_unit_GPU) + vox_size_x_GPU/2;
  int vy =  (int) floor((wy-min_y)/vox_unit_GPU);
  int vz =  (int) floor(wz/vox_unit_GPU) + vox_size_z_GPU/2;

  if( vx >= 0 && vx < vox_size_x_GPU && vy >= 0 && vy < vox_size_y_GPU && vz >= 0 && vz < vox_size_z_GPU){
      int vox_idx = vz * vox_size_x_GPU * vox_size_y_GPU * 12 + vy * vox_size_x_GPU * 12 + vx * 12 + lbl;

      atomicAdd(&one_hot_GPU[vox_idx], 1);

  } else {
    //printf("OUT OF VOX SPACE:%d rx: %2.2f ry: %2.2f rz: %2.2f  vx: %d vy: %d vz: %d\n",
    //        point_idx, wx, wy, wz, vx, vy, vz);
    return;
  }
}


__global__
void get_gt_kernel(int *one_hot_GPU, unsigned char * gt_grid_GPU, float *parameters_GPU){

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;

    int vox_size_x_GPU = (int)(parameters_GPU[VOX_SIZE_X]/2);
    int vox_size_y_GPU = (int)(parameters_GPU[VOX_SIZE_Y]/4);
    int vox_size_z_GPU = (int)(parameters_GPU[VOX_SIZE_Z]/2);

    if (vox_idx >= vox_size_x_GPU * vox_size_y_GPU * vox_size_z_GPU){
      return;
    }


    int z = (vox_idx / ( vox_size_x_GPU * vox_size_y_GPU))%vox_size_z_GPU ;
    int y = (vox_idx / vox_size_x_GPU) % vox_size_y_GPU;
    int x = vox_idx % vox_size_x_GPU;


    int lbl_count = 0;
    int occup_count = 0;
    unsigned char best_lbl = 0;

    for (int i=0; i<12; i++){
       occup_count += one_hot_GPU[vox_idx*12 + i];
       if (one_hot_GPU[vox_idx*12 + i]>lbl_count) {
           lbl_count = one_hot_GPU[vox_idx*12 + i];
           best_lbl = i;
       }
       //if (one_hot_GPU[vox_idx*12 + i] > 1) {
       //  printf("%d %d %d i:%d one_hot:%d \n", x, y, z, i, one_hot_GPU[vox_idx*12 + i]);
       //}
    }
    //if (best_lbl>0){
    //  printf("%d %d %d label %d \n", x, y, z, best_lbl);
    //}

    if (occup_count>32) { //reduce noise
      gt_grid_GPU[vox_idx] = best_lbl;
    }

}



void get_gt_CPP(float *point_cloud, int point_cloud_size, float *boundaries, unsigned char *gt_grid) {

  clock_tick t1 = start_timer();

  float *point_cloud_GPU;
  int *point_cloud_size_GPU;
  float *boundaries_GPU;
  int *one_hot_grid_GPU;
  unsigned char *gt_grid_GPU;

  int num_voxels = vox_size_x/2 * vox_size_y/4 * vox_size_z/2;

  if (debug==1) printf("get_gt - point_cloud_size: %d   voxel_size: %d %d %d\n" ,
                       point_cloud_size, vox_size_x/2 , vox_size_y/4 , vox_size_z/2);

  if (debug==1) printf("get_gt - boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
                       boundaries[0], boundaries[1], boundaries[2], boundaries[3], boundaries[4], boundaries[5]);

  if (debug==1) printf("(x %2.2f  y %2.2f z %2.2f l %2.2f\n" ,
                       point_cloud[0], point_cloud[1], point_cloud[2], point_cloud[3]);



  gpuErrchk(hipMalloc(&point_cloud_GPU, point_cloud_size * 4 * sizeof(float)));
  gpuErrchk(hipMalloc(&point_cloud_size_GPU, sizeof(int)));
  gpuErrchk(hipMalloc(&boundaries_GPU, 6 * sizeof(float)));
  gpuErrchk(hipMalloc(&one_hot_grid_GPU, num_voxels * 12 * sizeof(int)));
  gpuErrchk(hipMalloc(&gt_grid_GPU, num_voxels * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(point_cloud_GPU, point_cloud, point_cloud_size * 4 * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(point_cloud_size_GPU, &point_cloud_size, sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(boundaries_GPU, boundaries, 6 * sizeof(float), hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(gt_grid_GPU, 0, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMemset(one_hot_grid_GPU, 0, num_voxels * 12 * sizeof(int)));


  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((point_cloud_size + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("get_one_hot - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  get_one_hot_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(point_cloud_GPU, point_cloud_size_GPU,
                                                 boundaries_GPU, one_hot_grid_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  NUM_BLOCKS = int((num_voxels + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("get_gt - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  get_gt_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(one_hot_grid_GPU, gt_grid_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"get_gt duration");

  hipMemcpy(gt_grid, gt_grid_GPU,  num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(point_cloud_GPU);
  hipFree(point_cloud_size_GPU);
  hipFree(boundaries_GPU);
  hipFree(one_hot_grid_GPU);
  hipFree(gt_grid_GPU);

  end_timer(t1,"cleanup duration");

}

//Material downsampling
__global__
void downsample_grid_kernel_material( unsigned char *in_grid_GPU, unsigned char *out_grid_GPU, float *parameters_GPU) {

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
    float downscale = 4;

    int in_vox_size_x = (int)parameters_GPU[VOX_SIZE_X];
    int in_vox_size_y = (int)parameters_GPU[VOX_SIZE_Y];
    int in_vox_size_z = (int)parameters_GPU[VOX_SIZE_Z];
    int out_vox_size_x = (int)in_vox_size_x/downscale;
    int out_vox_size_y = (int)in_vox_size_y/downscale;
    int out_vox_size_z = (int)in_vox_size_z/downscale;

    if (vox_idx >= out_vox_size_x * out_vox_size_y * out_vox_size_z){
      return;
    }

    int z = (vox_idx / ( out_vox_size_x * out_vox_size_y))%out_vox_size_z ;
    int y = (vox_idx / out_vox_size_x) % out_vox_size_y;
    int x = vox_idx % out_vox_size_x;

    // for (int i = 0; i < 16; ++i) {
    //         sum_occupied[i] = 0;
    // }

    int sum_occupied[16] = {0};
    
  
    for (int tmp_x = x * downscale; tmp_x < (x + 1) * downscale; ++tmp_x) {
      for (int tmp_y = y * downscale; tmp_y < (y + 1) * downscale; ++tmp_y) {
        for (int tmp_z = z * downscale; tmp_z < (z + 1) * downscale; ++tmp_z) {

          int tmp_vox_idx = tmp_z * in_vox_size_x * in_vox_size_y + tmp_y * in_vox_size_z + tmp_x;

          // if (in_grid_GPU[tmp_vox_idx]> 0){
          //   sum_occupied += 1;
          //          }
          if (in_grid_GPU[tmp_vox_idx]!=0){
          
            if (in_grid_GPU[tmp_vox_idx]==47){
              sum_occupied[0] +=1;        
            }
            else if (in_grid_GPU[tmp_vox_idx]==193){
              sum_occupied[1] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==200){
              sum_occupied[2] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==23){
              sum_occupied[3] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==106){
              sum_occupied[4] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==88){
              sum_occupied[5] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==149){
              sum_occupied[6] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==191){
              sum_occupied[7] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==60){
              sum_occupied[8] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==102){
              sum_occupied[9] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==77){
              sum_occupied[10] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==22){
              sum_occupied[11] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==63){
              sum_occupied[12] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==169){
              sum_occupied[13] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==114){
              sum_occupied[14] += 1;
            }
            else if (in_grid_GPU[tmp_vox_idx]==119){
              sum_occupied[15] += 1;
            }
          }

        }
      }
    }


// __syncthreads(); // synchronize threads in the same block
// if (threadIdx.x == 0) { // let only one thread do the printing
//     for (int s=0; s<16; s++){
//       printf("sum occupied[%d]: %d\n", s, sum_occupied[s]);
//     }
// }
// __syncthreads();
int counter = 0;
__syncthreads(); // synchronize again before continuing
int maxIndex=0;
for (int i = 0; i < 16; i++) {
  if (sum_occupied[i] == 0){
    counter +=1;
  }
  if (sum_occupied[i] > sum_occupied[maxIndex]) {
    maxIndex = i;
  }
}
//printf("maxIndex is %d\n",maxIndex);
__syncthreads(); // synchronize again before continuing
  if (counter < 16){
    if (maxIndex==0){
      out_grid_GPU[vox_idx] = 47;        
    }
    else if (maxIndex==1){
      out_grid_GPU[vox_idx] = 193;
    }
    else if (maxIndex==2){
      out_grid_GPU[vox_idx] = 200;
    }
    else if (maxIndex==3){
      out_grid_GPU[vox_idx] = 23;
    }
    else if (maxIndex==4){
      out_grid_GPU[vox_idx] = 106;
    }
    else if (maxIndex==5){
      out_grid_GPU[vox_idx] = 88;
    }
    else if (maxIndex==6){
      out_grid_GPU[vox_idx] = 149;
    }
    else if (maxIndex==7){
      out_grid_GPU[vox_idx] = 191;
    }
    else if (maxIndex==8){
      out_grid_GPU[vox_idx] = 60;
    }
    else if (maxIndex==9){
      out_grid_GPU[vox_idx] = 102;
    }
    else if (maxIndex==10){
      out_grid_GPU[vox_idx] = 77;
    }
    else if (maxIndex==11){
      out_grid_GPU[vox_idx] = 22;
    }
    else if (maxIndex==12){
      out_grid_GPU[vox_idx] = 63;
    }
    else if (maxIndex==13){
      out_grid_GPU[vox_idx] = 169;
    }
    else if (maxIndex==14){
      out_grid_GPU[vox_idx] = 114;
    }
    else if (maxIndex==15){
      out_grid_GPU[vox_idx] = 119;
    }
  }
  else{
    out_grid_GPU[vox_idx] = 0;
  }


    // switch(maxIndex){
    //   case 0:
    //     out_grid_GPU[vox_idx] = 47;
    //     break;
    //   case 1:
    //       out_grid_GPU[vox_idx]= 193;
    //       break;
    //   case 2:
    //       out_grid_GPU[vox_idx]= 200;
    //       break;
    //   case 3:
    //       out_grid_GPU[vox_idx]= 23;
    //       break;
    //   case 4:
    //       out_grid_GPU[vox_idx]=196;
    //       break;
    //   case 5:
    //       out_grid_GPU[vox_idx]=41;
    //       break;
    //   case 6:
    //       out_grid_GPU[vox_idx]=149;
    //       break;
    //   case 7:
    //       out_grid_GPU[vox_idx]=191;
    //       break;
    //   case 8:
    //       out_grid_GPU[vox_idx]=60;
    //       break;
    //   case 9:
    //       out_grid_GPU[vox_idx]=102;
    //       break;
    //   case 10:
    //       out_grid_GPU[vox_idx]=77;
    //       break;
    //   case 11:
    //       out_grid_GPU[vox_idx]=22;
    //       break;
    //   case 12:
    //       out_grid_GPU[vox_idx]=63;
    //       break;
    //   case 13:
    //       out_grid_GPU[vox_idx]=169;
    //       break;
    //   case 14:
    //       out_grid_GPU[vox_idx]=114;
    //       break;
    //   case 15:
    //       out_grid_GPU[vox_idx]=119;
    //       break;
    //   default:
    //     //printf("Warning: Unexpected grayscale value %d\n", grayscale);
    //     break;
    //   }
   

}



__global__
void downsample_grid_kernel( unsigned char *in_grid_GPU, unsigned char *out_grid_GPU, float *parameters_GPU) {

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
    float downscale = 4;

    int in_vox_size_x = (int)parameters_GPU[VOX_SIZE_X];
    int in_vox_size_y = (int)parameters_GPU[VOX_SIZE_Y];
    int in_vox_size_z = (int)parameters_GPU[VOX_SIZE_Z];
    int out_vox_size_x = (int)in_vox_size_x/downscale;
    int out_vox_size_y = (int)in_vox_size_y/downscale;
    int out_vox_size_z = (int)in_vox_size_z/downscale;

    if (vox_idx >= out_vox_size_x * out_vox_size_y * out_vox_size_z){
      return;
    }

    int z = (vox_idx / ( out_vox_size_x * out_vox_size_y))%out_vox_size_z ;
    int y = (vox_idx / out_vox_size_x) % out_vox_size_y;
    int x = vox_idx % out_vox_size_x;

    int sum_occupied = 0;
  
    for (int tmp_x = x * downscale; tmp_x < (x + 1) * downscale; ++tmp_x) {
      for (int tmp_y = y * downscale; tmp_y < (y + 1) * downscale; ++tmp_y) {
        for (int tmp_z = z * downscale; tmp_z < (z + 1) * downscale; ++tmp_z) {

          int tmp_vox_idx = tmp_z * in_vox_size_x * in_vox_size_y + tmp_y * in_vox_size_z + tmp_x;

          if (in_grid_GPU[tmp_vox_idx]> 0){
            sum_occupied += 1;          }
        }
      }
    }
    if (sum_occupied>=4) {  //empty threshold
      out_grid_GPU[vox_idx] = 1;
    }    else {
    out_grid_GPU[vox_idx] = 0;
    }

}

__global__
void downsample_limits_kernel( unsigned char *in_grid_GPU, unsigned char *out_grid_GPU, float *parameters_GPU) {

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
    float downscale = 4;

    int in_vox_size_x = (int)parameters_GPU[VOX_SIZE_X];
    int in_vox_size_y = (int)parameters_GPU[VOX_SIZE_Y];
    int in_vox_size_z = (int)parameters_GPU[VOX_SIZE_Z];
    int out_vox_size_x = (int)in_vox_size_x/downscale;
    int out_vox_size_y = (int)in_vox_size_y/downscale;
    int out_vox_size_z = (int)in_vox_size_z/downscale;

    if (vox_idx >= out_vox_size_x * out_vox_size_y * out_vox_size_z){
      return;
    }

    int z = (vox_idx / ( out_vox_size_x * out_vox_size_y))%out_vox_size_z ;
    int y = (vox_idx / out_vox_size_x) % out_vox_size_y;
    int x = vox_idx % out_vox_size_x;

    int sum_occupied = 0;
    int sum_occluded = 0;

    for (int tmp_x = x * downscale; tmp_x < (x + 1) * downscale; ++tmp_x) {
      for (int tmp_y = y * downscale; tmp_y < (y + 1) * downscale; ++tmp_y) {
        for (int tmp_z = z * downscale; tmp_z < (z + 1) * downscale; ++tmp_z) {

          int tmp_vox_idx = tmp_z * in_vox_size_x * in_vox_size_y + tmp_y * in_vox_size_z + tmp_x;

          if (in_grid_GPU[tmp_vox_idx] == OCCUPIED){
            sum_occupied += 1;          }
          if (in_grid_GPU[tmp_vox_idx] == OCCLUDED){
            sum_occluded += 1;          }
        }
      }
    }
    if (sum_occupied + sum_occluded>=4) {  //empty threshold
      out_grid_GPU[vox_idx] = 1;
    }    else {
      out_grid_GPU[vox_idx] = 0;
    }

}

void downsample_material_grid_CPP(unsigned char *vox_grid, unsigned char *vox_grid_down) {

  clock_tick t1 = start_timer();

  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_down_GPU;

  //int* sum_occupied;

  int num_voxels = vox_size_x * vox_size_y * vox_size_z;
  int num_voxels_down = num_voxels/64;

  gpuErrchk(hipMalloc(&vox_grid_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_down_GPU, num_voxels_down * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(vox_grid_GPU, vox_grid, num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(vox_grid_down_GPU, 0, num_voxels_down * sizeof(unsigned char)));

  //gpuErrchk(cudaMalloc(&sum_occupied, 16 * sizeof(int)));


  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("downsample - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  downsample_grid_kernel_material<<<NUM_BLOCKS, NUM_THREADS>>>(vox_grid_GPU, vox_grid_down_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"downsample duration");

  hipMemcpy(vox_grid_down, vox_grid_down_GPU,  num_voxels_down * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(vox_grid_GPU);
  hipFree(vox_grid_down_GPU);
  //cudaFree(sum_occupied);

  end_timer(t1,"cleanup duration");

}


void downsample_grid_CPP(unsigned char *vox_grid, unsigned char *vox_grid_down) {

  clock_tick t1 = start_timer();

  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_down_GPU;

  int num_voxels = vox_size_x * vox_size_y * vox_size_z;
  int num_voxels_down = num_voxels/64;

  gpuErrchk(hipMalloc(&vox_grid_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_down_GPU, num_voxels_down * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(vox_grid_GPU, vox_grid, num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(vox_grid_down_GPU, 0, num_voxels_down * sizeof(unsigned char)));


  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("downsample - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  downsample_grid_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(vox_grid_GPU, vox_grid_down_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"downsample duration");

  hipMemcpy(vox_grid_down, vox_grid_down_GPU,  num_voxels_down * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(vox_grid_GPU);
  hipFree(vox_grid_down_GPU);

  end_timer(t1,"cleanup duration");

}


void downsample_limits_CPP(unsigned char *vox_grid, unsigned char *vox_grid_down) {

  clock_tick t1 = start_timer();

  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_down_GPU;

  int num_voxels = vox_size_x * vox_size_y * vox_size_z;
  int num_voxels_down = num_voxels/64;

  gpuErrchk(hipMalloc(&vox_grid_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_down_GPU, num_voxels_down * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(vox_grid_GPU, vox_grid, num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(vox_grid_down_GPU, 0, num_voxels_down * sizeof(unsigned char)));


  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("downsample - NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  downsample_limits_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(vox_grid_GPU, vox_grid_down_GPU, parameters_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"downsample duration");

  hipMemcpy(vox_grid_down, vox_grid_down_GPU,  num_voxels_down * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(vox_grid_GPU);
  hipFree(vox_grid_down_GPU);

  end_timer(t1,"cleanup duration");

}

__global__
void SquaredDistanceTransform(unsigned char *depth_data, unsigned char *vox_grid,
                              float *vox_tsdf, unsigned char *vox_limits, float *baseline,
                              int *width, int *height, float *boundaries_GPU, int *vol_number, float *parameters_GPU,unsigned char *material_data, unsigned char *material_map ) {

  float vox_unit_GPU = parameters_GPU[VOX_UNIT];
  float vox_margin_GPU = parameters_GPU[VOX_MARGIN];
  float sensor_w_GPU = parameters_GPU[SENSOR_W];
  float sensor_h_GPU = parameters_GPU[SENSOR_H];
  float f_GPU = parameters_GPU[CAM_F];
  int vox_size_x_GPU = (int)parameters_GPU[VOX_SIZE_X];
  int vox_size_y_GPU = (int)parameters_GPU[VOX_SIZE_Y];
  int vox_size_z_GPU = (int)parameters_GPU[VOX_SIZE_Z];

  float  min_x = boundaries_GPU[0];
  float  max_x = boundaries_GPU[1];
  float  min_y = boundaries_GPU[2];
  float  max_y = boundaries_GPU[3];
  float  min_z = boundaries_GPU[4];
  float  max_z = boundaries_GPU[5];

  int search_region = (int)roundf(vox_margin_GPU/vox_unit_GPU);

  int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (vox_idx >= vox_size_x_GPU * vox_size_y_GPU * vox_size_z_GPU){
      return;
  }

  int z = (vox_idx / ( vox_size_x_GPU * vox_size_y_GPU))%vox_size_z_GPU ;
  int y = (vox_idx / vox_size_x_GPU) % vox_size_y_GPU;
  int x = vox_idx % vox_size_x_GPU;


  // Get point in world coordinates XYZ -> YZX
  float wz;
  float wx;
  float wy;

  if (*vol_number == 1) {
      // Get point in world coordinates XYZ -> YZX

      wz = (float(z)) * vox_unit_GPU;                    //point_base[0]
      wx = (float(x)-vox_size_z_GPU/2) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 2) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(z) * vox_unit_GPU;                    //point_base[0]
      wx = float(x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (wx + wz) * (wx + wz))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz)); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }

  if (*vol_number == 3) {
      // Get point in world coordinates XYZ -> YZX
      wz = (float(z)-vox_size_x_GPU/2) * vox_unit_GPU;                    //point_base[0]
      wx = (float(x)) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 4) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(-x) * vox_unit_GPU;                    //point_base[0]
      wx = float(z) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));

      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;


      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }

  if (*vol_number == 5) {
      // Get point in world coordinates XYZ -> YZX
      wz = -(float(z)) * vox_unit_GPU;                    //point_base[0]
      wx = -(float(x)-vox_size_x_GPU/2) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }
  if (*vol_number == 6) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(-z) * vox_unit_GPU;                    //point_base[0]
      wx = float(-x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz))); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }
  if (*vol_number == 7) {
      // Get point in world coordinates XYZ -> YZX
      wz = -(float(z)-vox_size_x_GPU/2) * vox_unit_GPU;                    //point_base[0]
      wx = -(float(x)) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_x = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_y = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_x) || (abs(wy - min_y - CAM_HEIGHT)>fov_y)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 8) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(z) * vox_unit_GPU;                    //point_base[0]
      wx = float(-x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz)); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }



  if (wx==0.0 && wy==0 && wz==0){
    vox_tsdf[vox_idx] = 2000.;
    vox_limits[vox_idx] = OUT_OF_FOV;
    return;
  }

  float CV_PI = 3.141592;
  float longitude, latitude, point_depth, angle_disp;

  float hip1 = sqrtf(wx*wx + wz*wz);
  float hip2 = sqrtf(hip1*hip1 + wy*wy);

  float teta1, teta2;

  /*
  if (wy>0)
     teta1 = asin(wy/hip2);
  else
     teta1 = CV_PI - asin(wy/hip2);
  */
  teta1 = asin(wy/hip2);

  latitude = CV_PI/2 - teta1;

  //longitude = 3*CV_PI/2 - teta2;
  if (wx<0)
     teta2 = asin(wz/hip1);
  else
     teta2 = CV_PI - asin(wz/hip1);

  longitude = teta2;

  float  	unit_h, unit_w;	//angular size of 1 pixel
  float		disp_scale = 2;
  float		disp_offset = -120;
  int		max_radius = 30;


  unit_h = 1.0 / (*height);
  unit_w = 2.0 / (*width);

  int pixel_y = latitude /(unit_h * CV_PI);
  int pixel_x = longitude /(unit_w * CV_PI);

  int point_disparity = depth_data[pixel_y * *width + pixel_x];
  int point_material = material_data[pixel_y * *width + pixel_x];
  
  
  //printf("point_material -> %f,material_map value = %f,  Thread %d", point_material, material_map[vox_idx], vox_idx);

  if (point_disparity == 0){ // mising depth
      vox_tsdf[vox_idx] = -1.0;
      return;
  }

  angle_disp = (point_disparity / disp_scale + disp_offset) * unit_h * CV_PI;

  if (latitude + angle_disp <0)
    angle_disp = 0.01;

  if (angle_disp == 0)   {
	point_depth = max_radius;
	point_disparity = 0;
  }	else
    point_depth = *baseline / ((sin(latitude) / tan(latitude + angle_disp)) - cos(latitude));

  if (point_depth > max_radius || point_depth < 0.0) 	{
	point_depth = max_radius;
	point_disparity = 0;
  }

  float vox_depth =hip2;

  //OCCUPIED
  if (vox_grid[vox_idx] >0 ){
     vox_tsdf[vox_idx] = 0;
     vox_limits[vox_idx] = OCCUPIED;
     material_map[vox_idx] = point_material;
     //printf("Occupied, Thread %d, material_map value = %f\n", vox_idx, material_map[vox_idx]);
     return;
  }


  float sign;
  if (abs(point_depth - vox_depth) < 0.001){
      sign = -1; // avoid NaN
  }else{
      sign = (point_depth - vox_depth)/abs(point_depth - vox_depth);
  }
  vox_tsdf[vox_idx] = sign;
  //material_map[vox_idx] = point_material;
  //printf("Non occupied, Thread %d, material_map value = %f\n", vox_idx, material_map[vox_idx]);
  if (sign >0.0) {
    vox_limits[vox_idx] = EMPTY_VISIBLE;
  } else {
    vox_limits[vox_idx] = OCCLUDED;

  }
    int radius=search_region; // out -> in
    int found = 0;
    //fixed y planes
    int iiy = max(0,y-radius);
    for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iiy = min(y+radius,vox_size_y_GPU);
    for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed x planes
    int iix = max(0,x-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iix = min(x+radius,vox_size_x_GPU);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed z planes
    int iiz = max(0,z-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iiz = min(z+radius,vox_size_z_GPU);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }

    if (found == 0)
        return;

    radius=1; // in -> out
    found = 0;
    while (radius < search_region) {
        //fixed y planes
        int iiy = max(0,y-radius);
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiy = min(y+radius,vox_size_y_GPU);
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed x planes
        int iix = max(0,x-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iix = min(x+radius,vox_size_x_GPU);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed z planes
        int iiz = max(0,z-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiz = min(z+radius,vox_size_z_GPU);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        if (found == 1)
          return;

        radius++;

    }
}

__global__
void SquaredDistanceTransform_stanford(uint16_t *depth_data, unsigned char *vox_grid,
                              float *vox_tsdf, unsigned char *vox_limits,
                              int *width, int *height, float *boundaries_GPU, int *vol_number, float *parameters_GPU,
                              uint16_t *material_data, unsigned char *material_map ) {




  float vox_unit_GPU = parameters_GPU[VOX_UNIT];
  float vox_margin_GPU = parameters_GPU[VOX_MARGIN];
  float sensor_w_GPU = parameters_GPU[SENSOR_W];
  float sensor_h_GPU = parameters_GPU[SENSOR_H];
  float f_GPU = parameters_GPU[CAM_F];
  int vox_size_x_GPU = (int)parameters_GPU[VOX_SIZE_X];
  int vox_size_y_GPU = (int)parameters_GPU[VOX_SIZE_Y];
  int vox_size_z_GPU = (int)parameters_GPU[VOX_SIZE_Z];

  float  min_x = boundaries_GPU[0];
  float  max_x = boundaries_GPU[1];
  float  min_y = boundaries_GPU[2];
  float  max_y = boundaries_GPU[3];
  float  min_z = boundaries_GPU[4];
  float  max_z = boundaries_GPU[5];


  int search_region = (int)roundf(vox_margin_GPU/vox_unit_GPU);

  int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;


  if (vox_idx >= vox_size_x_GPU * vox_size_y_GPU * vox_size_z_GPU){
      return;
  }

  int z = (vox_idx / ( vox_size_x_GPU * vox_size_y_GPU))%vox_size_z_GPU ;
  int y = (vox_idx / vox_size_x_GPU) % vox_size_y_GPU;
  int x = vox_idx % vox_size_x_GPU;


  // Get point in world coordinates XYZ -> YZX
  float wz;
  float wx;
  float wy;

  if (*vol_number == 1) {
      // Get point in world coordinates XYZ -> YZX

      wz = (float(z)) * vox_unit_GPU;                    //point_base[0]
      wx = (float(x)-vox_size_z_GPU/2) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 2) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(z) * vox_unit_GPU;                    //point_base[0]
      wx = float(x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (wx + wz) * (wx + wz))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz)); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }

  if (*vol_number == 3) {
      // Get point in world coordinates XYZ -> YZX
      wz = (float(z)-vox_size_x_GPU/2) * vox_unit_GPU;                    //point_base[0]
      wx = (float(x)) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 4) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(-x) * vox_unit_GPU;                    //point_base[0]
      wx = float(z) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz)));

      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;


      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }

  if (*vol_number == 5) {
      // Get point in world coordinates XYZ -> YZX
      wz = -(float(z)) * vox_unit_GPU;                    //point_base[0]
      wx = -(float(x)-vox_size_x_GPU/2) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_w = (abs(wz)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (abs(wz)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wx)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }
  if (*vol_number == 6) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(-z) * vox_unit_GPU;                    //point_base[0]
      wx = float(-x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(abs(wx),abs(wz))* MIN(abs(wx),abs(wz))); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }
  if (*vol_number == 7) {
      // Get point in world coordinates XYZ -> YZX
      wz = -(float(z)-vox_size_x_GPU/2) * vox_unit_GPU;                    //point_base[0]
      wx = -(float(x)) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      //Calculating FOV
      float fov_x = (abs(wx)+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_y = (abs(wx)+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      if ((abs(wz)>fov_x) || (abs(wy - min_y - CAM_HEIGHT)>fov_y)) {
         //outside FOV
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }
  }

  if (*vol_number == 8) {  //Vol 1 has no adjustments

      // Get point in world coordinates XYZ -> YZX
      wz = float(z) * vox_unit_GPU;                    //point_base[0]
      wx = float(-x) * vox_unit_GPU; //point_base[1]
      wy = float(y) * vox_unit_GPU + (min_y-FLOOR_OFFSET);            //point_base[2]

      if (wx < min_x || wx > max_x || wy < min_y || wy > max_y || wz < min_z || wz > max_z ){
        // outside ROOM
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_ROOM;
        return;
      }

      float d = sqrtf(2.0 * (abs(wx) + abs(wz)) * (abs(wx) + abs(wz)))/2.0; //perpendicular distance to sensor plane
      float dc = d - sqrtf(2 * MIN(wx,wz)* MIN(wx,wz)); //distance to the center


      //Calculating FOV
      float fov_w = (d+GO_BACK) * (sensor_w_GPU/2)/f_GPU;
      float fov_h = (d+GO_BACK) * (sensor_h_GPU/2)/f_GPU;

      //if ((y<10) && ((x==20 && z==20)||(x==40 && z==40)||(x==180 && z==180)||(x==200 && z==200)||(x==0 && z==239)||(x==239 && z==0))) {
      //    printf("\n(%d,%d,%d) wx:%2.2f wy%2.2f wz:%2.2f d:%2.2f dc:%2.2f fov_w:%2.2f fov_h:%2.2f\n",
      //           x,y,z, wx,wy,wz,d,dc,fov_w,fov_h);
      //}

      if ((abs(dc)>fov_w) || (abs(wy - min_y - CAM_HEIGHT)>fov_h)) {
        vox_tsdf[vox_idx] = 2000.;
        vox_limits[vox_idx] = OUT_OF_FOV;
        return;
      }

  }



  if (wx==0.0 && wy==0 && wz==0){
    vox_tsdf[vox_idx] = 2000.;
    vox_limits[vox_idx] = OUT_OF_FOV;
    return;
  }

  float CV_PI = 3.141592;
  float longitude, latitude, point_depth, point_material;

  float hip1 = sqrtf(wx*wx + wz*wz);
  float hip2 = sqrtf(hip1*hip1 + wy*wy);

  float teta1, teta2;



  /*
  if (wy>0)
     teta1 = asin(wy/hip2);
  else
     teta1 = CV_PI - asin(wy/hip2);
  */
  teta1 = asin(wy/hip2);

  latitude = CV_PI/2 - teta1;

  //longitude = 3*CV_PI/2 - teta2;
  if (wx<0)
     teta2 = asin(wz/hip1);
  else
     teta2 = CV_PI - asin(wz/hip1);

  longitude = teta2;

  float  	unit_h, unit_w;	//angular size of 1 pixel
  int		max_radius = 30;


  unit_h = 1.0 / (*height);
  unit_w = 2.0 / (*width);

  int pixel_y = latitude /(unit_h * CV_PI);
  int pixel_x = longitude /(unit_w * CV_PI);

  point_depth = depth_data[pixel_y * *width + pixel_x];
  point_material = material_data[pixel_y * *width + pixel_x];

  if (point_depth == 65535.){ // mising depth
      vox_tsdf[vox_idx] = -1.0;
      return;
  }

  point_depth = point_depth /512.;

  if (point_depth > max_radius || point_depth < 0.0) 	{
	point_depth = max_radius;
  }

  float vox_depth =hip2;

  //OCCUPIED
  if (vox_grid[vox_idx] >0 ){
     vox_tsdf[vox_idx] = 0;
     material_map[vox_idx] = point_material;
     vox_limits[vox_idx] = OCCUPIED;
     return;
  }


  float sign;
  if (abs(point_depth - vox_depth) < 0.001){
      sign = -1; // avoid NaN
  }else{
      sign = (point_depth - vox_depth)/abs(point_depth - vox_depth);
  }
  vox_tsdf[vox_idx] = sign;
  if (sign >0.0) {
    vox_limits[vox_idx] = EMPTY_VISIBLE;
  } else {
    vox_limits[vox_idx] = OCCLUDED;

  }
    int radius=search_region; // out -> in
    int found = 0;
    //fixed y planes
    int iiy = max(0,y-radius);
    for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    
    iiy = min(y+radius,vox_size_y_GPU);
    for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed x planes
    int iix = max(0,x-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iix = min(x+radius,vox_size_x_GPU);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed z planes
    int iiz = max(0,z-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iiz = min(z+radius,vox_size_z_GPU);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }

    if (found == 0)
        return;

    radius=1; // in -> out
    found = 0;
    while (radius < search_region) {
        //fixed y planes
        int iiy = max(0,y-radius);
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiy = min(y+radius,vox_size_y_GPU);
        for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed x planes
        int iix = max(0,x-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iix = min(x+radius,vox_size_x_GPU);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size_z_GPU,z+radius+1); iiz++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed z planes
        int iiz = max(0,z-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiz = min(z+radius,vox_size_z_GPU);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size_y_GPU,y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size_x_GPU,x+radius+1); iix++){
                int iidx = iiz * vox_size_x_GPU * vox_size_y_GPU + iiy * vox_size_x_GPU + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        if (found == 1)
          return;

        radius++;

    }
}


void FlipTSDF_CPP(float *vox_tsdf){

  clock_tick t1 = start_timer();

  for (int vox_idx=0; vox_idx< vox_size_x*vox_size_y*vox_size_x; vox_idx++) {

      float value = float(vox_tsdf[vox_idx]);
      if (value > 1)
          value =1;


      float sign;
      if (abs(value) < 0.001)
        sign = 1;
      else
        sign = value/abs(value);

      vox_tsdf[vox_idx] = sign*(max(0.001,(1.0-abs(value))));
  }
  end_timer(t1,"FlipTSDF");
}


void FTSDFDepth_CPP(unsigned char *depth_data,
                      unsigned char *vox_grid,
                      unsigned char *vox_grid_edges,
                      float *vox_tsdf,
                      float *vox_tsdf_edges,
                      unsigned char *vox_limits,
                      float baseline,
                      int width,
                      int height,
                      float *boundaries,
                      int vol_number,
                      unsigned char *material_data,
                      unsigned char *material_map) {

  clock_tick t1 = start_timer();

  float         *boundaries_GPU;
  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_edges_GPU;
  unsigned char *depth_data_GPU;
  float         *vox_tsdf_GPU;
  float         *vox_tsdf_edges_GPU;
  unsigned char *vox_limits_GPU;
  float         *baseline_GPU;
  int           *width_GPU;
  int           *height_GPU;
  int           *vol_number_GPU;
  
  //Sourish Added this
  unsigned char *material_data_GPU;
  unsigned char *material_map_GPU;

  int num_voxels = vox_size_x * vox_size_y * vox_size_z;
  int num_pixels = width * height;

  if (debug==1) printf("FTSDFDepth - boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
                       boundaries[0], boundaries[1], boundaries[2], boundaries[3], boundaries[4], boundaries[5]);

  gpuErrchk(hipMalloc(&boundaries_GPU, 6 * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_grid_GPU,   num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_edges_GPU,   num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&depth_data_GPU, num_pixels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_tsdf_GPU,   num_voxels * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_tsdf_edges_GPU,   num_voxels * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_limits_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&baseline_GPU,   sizeof(float)));
  gpuErrchk(hipMalloc(&width_GPU,      sizeof(int)));
  gpuErrchk(hipMalloc(&height_GPU,     sizeof(int)));
  gpuErrchk(hipMalloc(&vol_number_GPU, sizeof(int)));
  //Sourish Added this
  gpuErrchk(hipMalloc(&material_data_GPU, num_pixels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&material_map_GPU,   num_voxels * sizeof(unsigned char)));

  gpuErrchk(hipMemcpy(boundaries_GPU,       boundaries,     6 * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vox_grid_GPU,         vox_grid,       num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vox_grid_edges_GPU,   vox_grid_edges, num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(depth_data_GPU,       depth_data,     num_pixels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(baseline_GPU,         &baseline,      sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(width_GPU,            &width,         sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(height_GPU,           &height,        sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vol_number_GPU,       &vol_number,    sizeof(int), hipMemcpyHostToDevice));
  //Sourish Added this
  gpuErrchk(hipMemcpy(material_data_GPU,      material_data,     num_pixels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(material_map_GPU,        material_map,       num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));


  gpuErrchk(hipMemset(vox_limits_GPU,       0,             num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMemset(vox_tsdf_GPU,         0,             num_voxels * sizeof(float)));
  gpuErrchk(hipMemset(vox_tsdf_edges_GPU,         0,             num_voxels * sizeof(float)));
  
 

  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((num_voxels + size_t(NUM_THREADS) - 1) / NUM_THREADS);

 /*  SquaredDistanceTransform(unsigned char *depth_data, float *vox_grid,
                              float *vox_tsdf, unsigned char *vox_limits, float *baseline,
                              int *width, int *height, float *boundaries_GPU, int *vol_number, float *parameters_GPU)
*/
  SquaredDistanceTransform<<<NUM_BLOCKS, NUM_THREADS>>>(depth_data_GPU, vox_grid_GPU, vox_tsdf_GPU, vox_limits_GPU,
                                                 baseline_GPU, width_GPU, height_GPU,
                                                 boundaries_GPU, vol_number_GPU, parameters_GPU, material_data_GPU,material_map_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  SquaredDistanceTransform<<<NUM_BLOCKS, NUM_THREADS>>>(depth_data_GPU, vox_grid_edges_GPU, vox_tsdf_edges_GPU, vox_limits_GPU,
                                                 baseline_GPU, width_GPU, height_GPU,
                                                 boundaries_GPU, vol_number_GPU, parameters_GPU, material_data_GPU,material_map_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"SquaredDistanceTransform duration");



  hipMemcpy(vox_tsdf, vox_tsdf_GPU,      num_voxels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vox_tsdf_edges, vox_tsdf_edges_GPU,      num_voxels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vox_limits, vox_limits_GPU,  num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);
  //Sourish Added this
  hipMemcpy(material_map,       material_map_GPU,       num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);

  // int i =0;
  // for (i=0; i < num_voxels * sizeof(unsigned char);i++){
  //   printf("material_map value = %d",material_map[i]);
  // }
  hipFree(boundaries_GPU);
  hipFree(vox_grid_GPU);
  hipFree(vox_grid_edges_GPU);
  hipFree(depth_data_GPU);
  hipFree(vox_tsdf_GPU);
  hipFree(vox_tsdf_edges_GPU);
  hipFree(vox_limits_GPU);
  hipFree(baseline_GPU);
  hipFree(width_GPU);
  hipFree(height_GPU);
  hipFree(vol_number_GPU);
  //Sourish Added this
  hipFree(material_data_GPU);
  hipFree(material_map_GPU);

  end_timer(t1,"cleanup duration");

  FlipTSDF_CPP(vox_tsdf);
  FlipTSDF_CPP(vox_tsdf_edges);

}

void FTSDFDepth_stanford_CPP(uint16_t *depth_data,
                      unsigned char *vox_grid,
                      unsigned char *vox_grid_edges,
                      float *vox_tsdf,
                      float *vox_tsdf_edges,
                      unsigned char *vox_limits,
                      int width,
                      int height,
                      float *boundaries,
                      int vol_number,
                      uint16_t *material_data,
                      unsigned char *material_map ) {

  clock_tick t1 = start_timer();

  float         *boundaries_GPU;
  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_edges_GPU;
  uint16_t *depth_data_GPU;
  float         *vox_tsdf_GPU;
  float         *vox_tsdf_edges_GPU;
  unsigned char *vox_limits_GPU;
  int           *width_GPU;
  int           *height_GPU;
  int           *vol_number_GPU;

  //Sourish Added this
  uint16_t *material_data_GPU;
  unsigned char *material_map_GPU;


  int num_voxels = vox_size_x * vox_size_y * vox_size_z;
  int num_pixels = width * height;

  if (debug==1) printf("FTSDFDepth - boundaries: (%2.2f %2.2f) (%2.2f %2.2f) (%2.2f %2.2f)\n" ,
                       boundaries[0], boundaries[1], boundaries[2], boundaries[3], boundaries[4], boundaries[5]);

  gpuErrchk(hipMalloc(&boundaries_GPU,     6 * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_grid_GPU,       num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&vox_grid_edges_GPU, num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&depth_data_GPU,     num_pixels * sizeof(uint16_t)));
  gpuErrchk(hipMalloc(&vox_tsdf_GPU,       num_voxels * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_tsdf_edges_GPU, num_voxels * sizeof(float)));
  gpuErrchk(hipMalloc(&vox_limits_GPU,     num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMalloc(&width_GPU,          sizeof(int)));
  gpuErrchk(hipMalloc(&height_GPU,         sizeof(int)));
  gpuErrchk(hipMalloc(&vol_number_GPU,     sizeof(int)));

  gpuErrchk(hipMemcpy(boundaries_GPU,      boundaries,     6 * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vox_grid_GPU,        vox_grid,       num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vox_grid_edges_GPU,  vox_grid_edges, num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(depth_data_GPU,      depth_data,     num_pixels * sizeof(uint16_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(width_GPU,           &width,         sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(height_GPU,          &height,        sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(vol_number_GPU,      &vol_number,    sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(vox_limits_GPU,       0,             num_voxels * sizeof(unsigned char)));
  gpuErrchk(hipMemset(vox_tsdf_GPU,         0,             num_voxels * sizeof(float)));
  gpuErrchk(hipMemset(vox_tsdf_edges_GPU,         0,             num_voxels * sizeof(float)));

  //Sourish Added this
  gpuErrchk(hipMemcpy(material_data_GPU,      material_data,     num_pixels * sizeof(uint16_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(material_map_GPU,        material_map,       num_voxels * sizeof(unsigned char), hipMemcpyHostToDevice))




  end_timer(t1, "Prepare duration");

  t1 = start_timer();
  int NUM_BLOCKS = int((num_voxels + size_t(NUM_THREADS) - 1) / NUM_THREADS);

 /*SquaredDistanceTransform(unsigned char *depth_data, float *vox_grid,
                              float *vox_tsdf, unsigned char *vox_limits, float *baseline,
                              int *width, int *height, float *boundaries_GPU, int *vol_number, float *parameters_GPU)
*/
  SquaredDistanceTransform_stanford<<<NUM_BLOCKS, NUM_THREADS>>>(depth_data_GPU, vox_grid_GPU, vox_tsdf_GPU, vox_limits_GPU,
                                                 width_GPU, height_GPU,
                                                 boundaries_GPU, vol_number_GPU, parameters_GPU, material_data_GPU,material_map_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  SquaredDistanceTransform_stanford<<<NUM_BLOCKS, NUM_THREADS>>>(depth_data_GPU, vox_grid_edges_GPU, vox_tsdf_edges_GPU, vox_limits_GPU,
                                                 width_GPU, height_GPU,
                                                 boundaries_GPU, vol_number_GPU, parameters_GPU, material_data_GPU,material_map_GPU);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );


  end_timer(t1,"SquaredDistanceTransform duration");

  hipMemcpy(vox_tsdf,       vox_tsdf_GPU,       num_voxels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vox_tsdf_edges, vox_tsdf_edges_GPU, num_voxels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vox_limits,     vox_limits_GPU,     num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);
  //Sourish Added this
  hipMemcpy(material_map,       material_map_GPU,       num_voxels * sizeof(unsigned char), hipMemcpyDeviceToHost);


  hipFree(boundaries_GPU);
  hipFree(vox_grid_GPU);
  hipFree(vox_grid_edges_GPU);
  hipFree(depth_data_GPU);
  hipFree(vox_tsdf_GPU);
  hipFree(vox_tsdf_edges_GPU);
  hipFree(vox_limits_GPU);
  hipFree(width_GPU);
  hipFree(height_GPU);
  hipFree(vol_number_GPU);
  //Sourish Added this
  hipFree(material_data_GPU);
  hipFree(material_map_GPU);

  end_timer(t1,"cleanup duration");

  FlipTSDF_CPP(vox_tsdf);
  FlipTSDF_CPP(vox_tsdf_edges);

}


extern "C" {
    void get_point_cloud  (float baseline,
                  unsigned char *depth_data,
                  float *point_cloud,
                  int width,
                  int height) {
                                 get_point_cloud_CPP  (baseline,
                                              depth_data,
                                              point_cloud,
                                              width,
                                              height) ;
                  }
    void get_point_cloud_stanford  (uint16_t *depth_data,
                  float *point_cloud,
                  int width,
                  int height) {
                                 get_point_cloud_stanford_CPP  (depth_data,
                                              point_cloud,
                                              width,
                                              height) ;
                  }


    void get_voxels (float *point_cloud,
                     unsigned char *edges_data,
                     int width, int height,
                     float *boundaries,
                     int vol_number,
                     unsigned char *vox_grid,
                     unsigned char *vox_grid_edges) {
                                 get_voxels_CPP(point_cloud,
                                                edges_data,
                                                width, height,
                                                boundaries,
                                                vol_number,
                                                vox_grid,
                                                vox_grid_edges) ;
                  }

    void get_gt (float *point_cloud,
                     int point_cloud_size,
                     float *boundaries,
                     unsigned char *gt_grid) {
                                 get_gt_CPP(point_cloud,
                                                point_cloud_size,
                                                boundaries,
                                                gt_grid) ;
                  }

    void FTSDFDepth(unsigned char *depth_data,
                      unsigned char *vox_grid,
                      unsigned char *vox_grid_edges,
                      float *vox_tsdf,
                      float *vox_tsdf_edges,
                      unsigned char *vox_limits,
                      float baseline,
                      int width,
                      int height,
                      float *boundaries,
                      int vol_number,
                      unsigned char *material_data,
                      unsigned char *material_map) {
                                 FTSDFDepth_CPP(depth_data,
                                                vox_grid,
                                                vox_grid_edges,
                                                vox_tsdf,
                                                vox_tsdf_edges,
                                                vox_limits,
                                                baseline,
                                                width,
                                                height,
                                                boundaries,
                                                vol_number,
                                                material_data,
                                                material_map);
                  }
    void FTSDFDepth_stanford(uint16_t *depth_data,
                      unsigned char *vox_grid,
                      unsigned char *vox_grid_edges,
                      float *vox_tsdf,
                      float *vox_tsdf_edges,
                      unsigned char *vox_limits,
                      int width,
                      int height,
                      float *boundaries,
                      int vol_number, 
                      uint16_t *material_data,
                      unsigned char *material_map) {
                                 FTSDFDepth_stanford_CPP(depth_data,
                                                vox_grid,
                                                vox_grid_edges,
                                                vox_tsdf,
                                                vox_tsdf_edges,
                                                vox_limits,
                                                width,
                                                height,
                                                boundaries,
                                                vol_number,
                                                material_data,
                                                material_map) ;
                  }

    void downsample_material_grid (unsigned char *vox_grid,
                            unsigned char *vox_grid_down) {
                                 downsample_material_grid_CPP(vox_grid,
                                                vox_grid_down) ;
                          }
    void downsample_grid (unsigned char *vox_grid,
                            unsigned char *vox_grid_down) {
                                 downsample_grid_CPP(vox_grid,
                                                vox_grid_down) ;
                  }
    void downsample_limits (unsigned char *vox_grid,
                            unsigned char *vox_grid_down) {
                                 downsample_limits_CPP(vox_grid,
                                                vox_grid_down) ;
                  }
    void setup(int device, int num_threads,
               float v_unit, float v_margin,
               float f, float sensor_w, float sensor_h,
               int vox_size_x, int vox_size_y, int vox_size_z,
               int debug_flag){
                                  setup_CPP(device, num_threads,
                                            v_unit, v_margin,
                                            f, sensor_w, sensor_h,
                                            vox_size_x, vox_size_y, vox_size_z,
                                            debug_flag);
                  }



    void finish(){
                                  clear_parameters_GPU();
    }
/*    void ProcessEdges(int *vox_size
                  int out_scale,
                  unsigned char *depth_data,
                  unsigned char *edges_data,
                  float *vox_tsdf,
                  float *vox_edges,
                  float *tsdf_edges,
                  float *vox_limits,
                  int *segmentation_label_downscale) {
                                 ProcessEdges_CPP(vox_size,
                                             out_scale,
                                             depth_data,
                                             edges_data,
                                             vox_tsdf,
                                             vox_edges,
                                             tsdf_edges,
                                             vox_limits,
                                             segmentation_label_downscale) ;
                  }

    void get_grid(float baseline, int *vox_size,
                  unsigned char *depth_data,
                  unsigned char *vox_grid) {
                                 get_grid_CPP(baseline, vox_size,
                                             depth_data,
                                             vox_grid) ;
                  }

    void get_rgb_grid(float baseline, int *vox_size,
                  unsigned char *depth_data,
                  unsigned char *rgb_data,
                  unsigned char *vox_grid) {
                                 get_rgb_grid_CPP(baseline, vox_size,
                                             depth_data,
                                             rgb_data,
                                             vox_grid) ;
                  }
*/
}



/*
void destroy_parameters_GPU(float *parameters_GPU){

  cudaFree(parameters_GPU);

}


__global__
void depth2Grid(float *baseline, int *vox_size,  unsigned char *depth_data,
                unsigned char *vox_grid, float *parameters_GPU){

  //Get Parameters
  int frame_width_GPU, frame_height_GPU, total_width_GPU;
  float vox_unit_GPU, vox_margin_GPU;

  get_parameters_GPU(parameters_GPU, &frame_width_GPU, &frame_height_GPU, &total_width_GPU,
                                     &vox_unit_GPU, &vox_margin_GPU);

  //if (threadIdx.x==0) printf("fwg %d  fwg %d", frame_width_GPU,frame_height_GPU);

  //Rerieve pixel coodinates
  int pixel_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (pixel_idx >= frame_width_GPU * frame_height_GPU)
    return;

  int pixel_y = pixel_idx / frame_width_GPU;
  int pixel_x = pixel_idx % frame_width_GPU;


  float     CV_PI = 3.141592;

  int		max_radius = 30;
  int		inf_border = 160;		// Range (in pixel) from the pole to exclude from point cloud generation
  double	unit_h, unit_w;	//angular size of 1 pixel
  float		disp_scale = 2;
  float		disp_offset = -120;



  unit_h = 1.0 / (frame_height_GPU);
  unit_w = 2.0 / (total_width_GPU);




  // Get point in world coordinate
  // Try to parallel later


  int point_disparity = depth_data[pixel_y * frame_width_GPU + pixel_x];

  if (point_disparity == 0)
	return;

  if (pixel_y<inf_border || pixel_y> frame_height_GPU - inf_border)
	return;

  float longitude, latitude, radius, angle_disp;

  latitude = pixel_y * unit_h * CV_PI;

  longitude = pixel_x * unit_w * CV_PI;

  angle_disp = (point_disparity / disp_scale + disp_offset) * unit_h * CV_PI;

  if (latitude + angle_disp <0)
    angle_disp = 0.01;

  if (angle_disp == 0)   {
	radius = max_radius;
	point_disparity = 0;
  }	else
	radius = *baseline / ((sin(latitude) / tan(latitude + angle_disp)) - cos(latitude));

  if (radius > max_radius || radius < 0.0) 	{
	radius = max_radius;
	point_disparity = 0;
  }

  //world coordinates
  float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  float ry = radius*sin(latitude)*sin(CV_PI - longitude);
  float rz = radius*cos(latitude);

  //voxel coordinates
  int z = (int)floor(rz / vox_unit_GPU + vox_size[2]/2);
  int x = (int)floor(rx / vox_unit_GPU + vox_size[0]/2);
  int y = (int)floor(ry / vox_unit_GPU);

  //too close
  if (z<.5)
    return;


  // mark vox_out with 1.0
  if( x >= 0 && x < vox_size[0] && y >= 0 && y < vox_size[1] && z >= 0 && z < vox_size[2]){
      int vox_idx = z * vox_size[0] * vox_size[1] + y * vox_size[0] + x;
      vox_grid[vox_idx] = float(1.0);
      //printf("OK idx:%d d:%d px:%d py:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n",
      //    pixel_idx, point_disparity, pixel_x, pixel_y, rx, ry, rz, x, y, z);
  } else {
          printf("OUT idx:%d d:%d px:%d py:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n",
          pixel_idx, point_disparity, pixel_x, pixel_y, rx, ry, rz, x, y, z);
  }
}




void get_grid_CPP(float baseline, int *vox_size, unsigned char *depth_data, unsigned char *vox_grid_down) {

  clock_tick t1 = start_timer();
  int num_voxels = vox_size[0] * vox_size[1] * vox_size[2];
  int vox_size_down[] = {vox_size[0]/4, vox_size[1]/4, vox_size[2]/4};
  int num_voxels_down = vox_size_down[0] * vox_size_down[1] * vox_size_down[2];

  float *baseline_GPU;
  unsigned char *depth_data_GPU;
  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_down_GPU;
  int *vox_size_GPU;
  int *vox_size_down_GPU;




  if (debug==1) printf("cudaMalloc1\n");
  gpuErrchk(cudaMalloc(&baseline_GPU, sizeof(float)));

  if (debug==1) printf("cudaMalloc2\n");
  gpuErrchk(cudaMalloc(&vox_size_GPU, 3 * sizeof(int)));

  if (debug==1) printf("cudaMallod depth_data_GPU\n");
  gpuErrchk(cudaMalloc(&depth_data_GPU, frame_height * frame_width * sizeof(unsigned char)));
  cudaMalloc(&vox_grid_GPU, num_voxels * sizeof(unsigned char));
  cudaMemset(vox_grid_GPU, 0, num_voxels * sizeof(unsigned char));

  cudaMemcpy(baseline_GPU, &baseline, sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpy(vox_size_GPU, vox_size, 3 * sizeof(int), cudaMemcpyHostToDevice);

  if (debug==1) printf("cudaMencpy depth_data_GPU\n");
  gpuErrchk(cudaMemcpy(depth_data_GPU, depth_data, frame_height * frame_width * sizeof(unsigned char), cudaMemcpyHostToDevice));

  end_timer(t1, "Prepare duration");

  if (debug==1) printf("frame width: %d   frame heigth: %d   num_voxels %d\n" , frame_width,frame_height, num_voxels);


  t1 = start_timer();
  // from depth map to binaray voxel representation
  //depth2Grid<<<frame_width,frame_height>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);


  int NUM_BLOCKS = int((frame_width*frame_height + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  depth2Grid<<<NUM_BLOCKS, NUM_THREADS>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
                                           vox_grid_GPU, parameters_GPU);
  //depth2Grid<<<3, 1024>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);

  if (debug==1) printf("depth2Grid\n");
  gpuErrchk( cudaPeekAtLastError() );

  if (debug==1) printf("cudaDeviceSynchronize\n");
  gpuErrchk( cudaDeviceSynchronize() );

  end_timer(t1,"depth2Grid duration");




  gpuErrchk(cudaMalloc(&vox_size_down_GPU, 3 * sizeof(int)));
  cudaMalloc(&vox_grid_down_GPU, num_voxels_down * sizeof(unsigned char));

  cudaMemcpy(vox_size_down_GPU, vox_size_down, 3 * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemset(vox_grid_down_GPU, 0, num_voxels_down * sizeof(unsigned char));

  NUM_BLOCKS = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  grid_downsample_Kernel<<<NUM_BLOCKS, NUM_THREADS>>>(vox_size_GPU,  vox_size_down_GPU,
                                           vox_grid_GPU, vox_grid_down_GPU);

  if (debug==1) printf("grid_downsample_Kernel\n");
  gpuErrchk( cudaPeekAtLastError() );


  t1 = start_timer();
  cudaMemcpy(vox_grid_down, vox_grid_down_GPU, num_voxels_down * sizeof(unsigned char), cudaMemcpyDeviceToHost);

  cudaFree(baseline_GPU);
  cudaFree(vox_size_GPU);
  cudaFree(depth_data_GPU);
  cudaFree(vox_grid_GPU);

  end_timer(t1,"closeup duration");

  if (debug==1) printf("0 %d\n", depth_data[0]);
  if (debug==1) printf("1 %d\n", depth_data[1]);
  if (debug==1) printf("2 %d\n", depth_data[2]);
  if (debug==1) printf("0fw %d\n", depth_data[0+frame_width]);
  if (debug==1) printf("1fw %d\n", depth_data[1+frame_width]);
  if (debug==1) printf("2fw %d\n", depth_data[2+frame_width]);
  if (debug==1) printf("02fw %d\n", depth_data[0+2*frame_width]);
  if (debug==1) printf("12fw %d\n", depth_data[1+2*frame_width]);
  if (debug==1) printf("22fw %d\n", depth_data[2+2*frame_width]);

}


__global__
void rgb2Grid(float *baseline, int *vox_size,  unsigned char *depth_data, unsigned char *rgb_data,
                unsigned char *vox_grid, float *parameters_GPU){

  //Get Parameters
  int frame_width_GPU, frame_height_GPU, total_width_GPU;
  float vox_unit_GPU, vox_margin_GPU;

  get_parameters_GPU(parameters_GPU, &frame_width_GPU, &frame_height_GPU, &total_width_GPU,
                                     &vox_unit_GPU, &vox_margin_GPU);

  //if (threadIdx.x==0) printf("fwg %d  fwg %d", frame_width_GPU,frame_height_GPU);

  //Rerieve pixel coodinates
  int pixel_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (pixel_idx >= frame_width_GPU * frame_height_GPU)
    return;

  int pixel_y = pixel_idx / frame_width_GPU;
  int pixel_x = pixel_idx % frame_width_GPU;


  float     CV_PI = 3.141592;

  int		max_radius = 30;
  int		inf_border = 160;		// Range (in pixel) from the pole to exclude from point cloud generation
  double	unit_h, unit_w;	//angular size of 1 pixel
  float		disp_scale = 2;
  float		disp_offset = -120;



  unit_h = 1.0 / (frame_height_GPU);
  unit_w = 2.0 / (total_width_GPU);




  // Get point in world coordinate
  // Try to parallel later


  int point_disparity = depth_data[pixel_y * frame_width_GPU + pixel_x];
  int point_r = rgb_data[3* (pixel_y * frame_width_GPU + pixel_x) + 0];
  int point_g = rgb_data[3* (pixel_y * frame_width_GPU + pixel_x) + 1];
  int point_b = rgb_data[3* (pixel_y * frame_width_GPU + pixel_x) + 2];

  if (point_disparity == 0)
	return;

  if (pixel_y<inf_border || pixel_y> frame_height_GPU - inf_border)
	return;

  float longitude, latitude, radius, angle_disp;

  latitude = pixel_y * unit_h * CV_PI;

  longitude = pixel_x * unit_w * CV_PI;

  angle_disp = (point_disparity / disp_scale + disp_offset) * unit_h * CV_PI;

  if (latitude + angle_disp <0)
    angle_disp = 0.01;

  if (angle_disp == 0)   {
	radius = max_radius;
	point_disparity = 0;
  }	else
	radius = *baseline / ((sin(latitude) / tan(latitude + angle_disp)) - cos(latitude));

  if (radius > max_radius || radius < 0.0) 	{
	radius = max_radius;
	point_disparity = 0;
  }

  //too close
  //if (latitude < CV_PI/4) || (latitude > CV_PI - CV_PI/4))
  if (latitude < CV_PI/3)
    return;


  //world coordinates
  //float rx = radius*sin(latitude)*cos(CV_PI - longitude);
  //float ry = radius*sin(latitude)*sin(CV_PI - longitude);
  //float rz = radius*cos(latitude);
  //voxel coordinates
  //int z = (int)floor(rz / vox_unit_GPU + vox_size[2]/2);
  //int x = (int)floor(rx / vox_unit_GPU + vox_size[0]/2));
  //int y = (int)floor(ry / vox_unit_GPU);


  float rx = -radius*sin(latitude)*cos(CV_PI - longitude);
  float rz = radius*sin(latitude)*sin(CV_PI - longitude);
  float ry = radius*cos(latitude) + 1.45 +0.20; //+.20cm to get the floor

  //voxel coordinates
  int z = (int)floor(rz / vox_unit_GPU);
  int x = (int)floor(rx / vox_unit_GPU);// + vox_size[0]/2);
  int y = (int)floor(ry / vox_unit_GPU);// + vox_size[1]/2);



  // mark vox_out with 1.0
  if( x >= 0 && x < vox_size[0] && y >= 0 && y < vox_size[1] && z >= 0 && z < vox_size[2]){
      int vox_idx = z * vox_size[0] * vox_size[1] + y * vox_size[0] + x;
      vox_grid[3 * vox_idx + 0] = point_r;
      vox_grid[3 * vox_idx + 1] = point_g;
      vox_grid[3 * vox_idx + 2] = point_b;
      //printf("RGB o.o:%d OK idx:%d d:%d px:%d py:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n",
      //    depth_data[0],pixel_idx, point_disparity, pixel_x, pixel_y, rx, ry, rz, x, y, z);
  } else {
          //printf("RGB OUT idx:%d d:%d px:%d py:%d rx:%f ry:%f rz:%f vx:%d vy:%d vz:%d\n",
          //pixel_idx, point_disparity, pixel_x, pixel_y, rx, ry, rz, x, y, z);
  }
}

__global__
void rgb_grid_downsample_Kernel( int *in_vox_size, int *out_vox_size,
                        unsigned char *in_grid_GPU, unsigned char *out_grid_GPU) {

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;


    if (vox_idx >= out_vox_size[0] * out_vox_size[1] * out_vox_size[2]){
      return;
    }

    float label_downscale = in_vox_size[0]/out_vox_size[0];

    //printf("down_size %d\n",down_size);

    int z = (vox_idx / ( out_vox_size[0] * out_vox_size[1]))%out_vox_size[2] ;
    int y = (vox_idx / out_vox_size[0]) % out_vox_size[1];
    int x = vox_idx % out_vox_size[0];

    int sum_occupied = 0;
    int r = 0;
    int g = 0;
    int b = 0;

    for (int tmp_x = x * label_downscale; tmp_x < (x + 1) * label_downscale; ++tmp_x) {
      for (int tmp_y = y * label_downscale; tmp_y < (y + 1) * label_downscale; ++tmp_y) {
        for (int tmp_z = z * label_downscale; tmp_z < (z + 1) * label_downscale; ++tmp_z) {

          int tmp_vox_idx = tmp_z * in_vox_size[0] * in_vox_size[1] + tmp_y * in_vox_size[0] + tmp_x;

          if (in_grid_GPU[3 * tmp_vox_idx + 0] +
              in_grid_GPU[3 * tmp_vox_idx + 1] +
              in_grid_GPU[3 * tmp_vox_idx + 2]> 0){
            sum_occupied += 1;
            r +=  in_grid_GPU[3 * tmp_vox_idx + 0];
            g +=  in_grid_GPU[3 * tmp_vox_idx + 1];
            b +=  in_grid_GPU[3 * tmp_vox_idx + 2];
          }
        }
      }
    }
    if (sum_occupied<8) {  //empty threshold
      out_grid_GPU[3 * vox_idx + 0 ] = 0;
      out_grid_GPU[3 * vox_idx + 1 ] = 0;
      out_grid_GPU[3 * vox_idx + 2 ] = 0;
    }else{
      out_grid_GPU[3 * vox_idx + 0 ] = r/sum_occupied;
      out_grid_GPU[3 * vox_idx + 1 ] = g/sum_occupied;
      out_grid_GPU[3 * vox_idx + 2 ] = b/sum_occupied;
    }
}


float mae_calc_y(int y, unsigned char *vox_grid, int *vox_size, int top){
  int n = 0;
  float mae = 0.0;
  for (int x=0; x<vox_size[0]; x++) {
    for (int z=0; z<vox_size[2]; z++) {
       if (top) {
         //from top to bottom
         for (int y_try=vox_size[1] - 1; y_try>=vox_size[1]/2; y_try--){
           int vox_idx_try = z * vox_size[0] * vox_size[1] + y_try * vox_size[0] + x;
           if (vox_grid[3*vox_idx_try + 0] +  vox_grid[3*vox_idx_try + 1] + vox_grid[3*vox_idx_try + 2] > 0){
             n++;
             mae += abs(y_try - y);
           }
         }
       } else {
         //from bottom to top
         for (int y_try=0; y_try< vox_size[1]/2; y_try++){
           int vox_idx_try = z * vox_size[0] * vox_size[1] + y_try * vox_size[0] + x;
           if (vox_grid[3*vox_idx_try + 0] +  vox_grid[3*vox_idx_try + 1] + vox_grid[3*vox_idx_try + 2] > 0){
             n++;
             mae += abs(y_try - y);
           }
         }
       }
    }

  }

  mae /= n;
  //printf("top:%d y:%d  n:%d mae:%f vox_size: %d %d %d\n",top, y,n, mae, vox_size[0], vox_size[1], vox_size[2]);
  return(mae);

}

void find_vox_limits_CPP(unsigned char *vox_grid, unsigned char *vox_limits, int *vox_size, unsigned char *depth_data){
  //Top
  float min_mae = -1.;
  float mae;
  unsigned char ceil_y= vox_size[1] - 1;
  int top = 1;
  for (int y=vox_size[1] - 1; y>=vox_size[1]/2; y--){
    mae = mae_calc_y(y, vox_grid, vox_size, top);
    if ((mae<min_mae) || (min_mae==-1.)){
      min_mae = mae;
      ceil_y = y;
    }
  }
  printf("Top = %d\n", ceil_y);

  //adjust_top(vox_grid, vox_limits, vox_size, depth_data, ceil_y)


  unsigned char floor_y= 0;
  top = 0;
  min_mae = -1.;
  for (int y=0; y<vox_size[1]/2; y++){
    mae = mae_calc_y(y, vox_grid, vox_size, top);
    if ((mae<min_mae) || (min_mae==-1.)){
      min_mae = mae;
      floor_y = y;
    }
  }
  printf("Bottom = %d\n", floor_y);

}
*/


/*
void get_rgb_grid_CPP(float baseline, int *vox_size, unsigned char *depth_data, unsigned char *rgb_data, unsigned char *vox_grid_down) {

  clock_tick t1 = start_timer();
  int num_voxels = vox_size[0] * vox_size[1] * vox_size[2];
  int vox_size_down[] = {vox_size[0]/4, vox_size[1]/4, vox_size[2]/4};
  int num_voxels_down = vox_size_down[0] * vox_size_down[1] * vox_size_down[2];

  float *baseline_GPU;
  unsigned char *depth_data_GPU;
  unsigned char *rgb_data_GPU;
  unsigned char *vox_grid_GPU;
  unsigned char *vox_grid_down_GPU;
  unsigned char *vox_grid;
  unsigned char *vox_limits;
  int *vox_size_GPU;
  int *vox_size_down_GPU;

  //for (int i=0; i< frame_width* 10; i++){
  //  if (depth_data[i] != 0) {

  //    printf("i:%d y%d x:%d val:%d\n", i, i/frame_width, i%frame_width, depth_data[i] );

  //  }
  //}



  //if (debug==1) printf("dd %d %d %d %d\n", depth_data[0],depth_data[1],depth_data[2],depth_data[3] );
  //if (debug==1) printf("dd %d %d %d\n", depth_data[0+ frame_width],depth_data[1+frame_width],depth_data[2+frame_width] );
  //if (debug==1) printf("baseline %f\n", baseline);
  //if (debug==1) printf("vox size %d %d %d \n", vox_size[0], vox_size[1], vox_size[2]);
  //if (debug==1) printf("fw %d 600-350 %d\n", frame_width, depth_data[350+600*frame_width]);
  //if (debug==1) printf("rgb data 600-350 %d %d %d\n", rgb_data[350+600*frame_width],
  //                                                    rgb_data[351+600*frame_width],
   //                                                   rgb_data[352+600*frame_width] );

  if (debug==1) printf("cudaMalloc1\n");
  gpuErrchk(cudaMalloc(&baseline_GPU, sizeof(float)));

  if (debug==1) printf("cudaMalloc2\n");
  gpuErrchk(cudaMalloc(&vox_size_GPU, 3 * sizeof(int)));

  if (debug==1) printf("cudaMallod depth_data_GPU\n");
  gpuErrchk(cudaMalloc(&depth_data_GPU, frame_height * frame_width * sizeof(unsigned char)));
  gpuErrchk(cudaMalloc(&rgb_data_GPU, 3 * frame_height * frame_width * sizeof(unsigned char)));



  cudaMalloc(&vox_grid_GPU, 3 * num_voxels * sizeof(unsigned char));
  cudaMemset(vox_grid_GPU, 0, 3 * num_voxels * sizeof(unsigned char));

  cudaMemcpy(baseline_GPU, &baseline, sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpy(vox_size_GPU, vox_size, 3 * sizeof(int), cudaMemcpyHostToDevice);

  if (debug==1) printf("cudaMencpy depth_data_GPU\n");
  gpuErrchk(cudaMemcpy(depth_data_GPU, depth_data, frame_height * frame_width * sizeof(unsigned char), cudaMemcpyHostToDevice));
  gpuErrchk(cudaMemcpy(rgb_data_GPU, rgb_data, 3 * frame_height * frame_width * sizeof(unsigned char), cudaMemcpyHostToDevice));

  end_timer(t1, "Prepare duration");

  if (debug==1) printf("frame width: %d   frame heigth: %d   num_voxels %d\n" , frame_width,frame_height, num_voxels);


  t1 = start_timer();
  // from depth map to binaray voxel representation
  //depth2Grid<<<frame_width,frame_height>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);


  int NUM_BLOCKS = int((frame_width*frame_height + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  rgb2Grid<<<NUM_BLOCKS, NUM_THREADS>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU, rgb_data_GPU,
                                           vox_grid_GPU, parameters_GPU);
  //depth2Grid<<<3, 1024>>>(baseline_GPU, vox_size_GPU,  depth_data_GPU,
  //                                         vox_grid_GPU, parameters_GPU);

  if (debug==1) printf("depth2Grid\n");
  gpuErrchk( cudaPeekAtLastError() );

  if (debug==1) printf("cudaDeviceSynchronize\n");
  gpuErrchk( cudaDeviceSynchronize() );

  end_timer(t1,"depth2Grid duration");

  vox_grid= (unsigned char *)malloc(num_voxels * 3 * sizeof(unsigned char));
  vox_limits= (unsigned char *)malloc(num_voxels * sizeof(unsigned char));

  cudaMemcpy(vox_grid, vox_grid_GPU, 3* num_voxels*sizeof(unsigned char), cudaMemcpyDeviceToHost);

  find_vox_limits_CPP(vox_grid, vox_limits, vox_size, depth_data);


  gpuErrchk(cudaMalloc(&vox_size_down_GPU, 3 * sizeof(int)));
  cudaMalloc(&vox_grid_down_GPU, 3 *num_voxels_down * sizeof(unsigned char));

  cudaMemcpy(vox_size_down_GPU, vox_size_down, 3 * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemset(vox_grid_down_GPU, 0, 3 * num_voxels_down * sizeof(unsigned char));

  NUM_BLOCKS = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  if (debug==1) printf("NUM_BLOCKS: %d   NUM_THREADS: %d\n" , NUM_BLOCKS,NUM_THREADS);

  rgb_grid_downsample_Kernel<<<NUM_BLOCKS, NUM_THREADS>>>(vox_size_GPU,  vox_size_down_GPU,
                                           vox_grid_GPU, vox_grid_down_GPU);

  if (debug==1) printf("grid_downsample_Kernel\n");
  gpuErrchk( cudaPeekAtLastError() );


  t1 = start_timer();
  cudaMemcpy(vox_grid_down, vox_grid_down_GPU, 3 * num_voxels_down * sizeof(unsigned char), cudaMemcpyDeviceToHost);

  cudaFree(baseline_GPU);
  cudaFree(vox_size_GPU);
  cudaFree(depth_data_GPU);
  cudaFree(vox_grid_GPU);

  end_timer(t1,"closeup duration");

  if (debug==1) printf("0 %d\n", depth_data[0]);
  if (debug==1) printf("1 %d\n", depth_data[1]);
  if (debug==1) printf("2 %d\n", depth_data[2]);
  if (debug==1) printf("0fw %d\n", depth_data[0+frame_width]);
  if (debug==1) printf("1fw %d\n", depth_data[1+frame_width]);
  if (debug==1) printf("2fw %d\n", depth_data[2+frame_width]);
  if (debug==1) printf("02fw %d\n", depth_data[0+2*frame_width]);
  if (debug==1) printf("12fw %d\n", depth_data[1+2*frame_width]);
  if (debug==1) printf("600-350 %d\n", depth_data[350+600*frame_width]);

}

*/

/*




__device__
float modeLargerZero(const int *values, int size) {
  int count_vector[NUM_CLASSES] = {0};

  for (int i = 0; i < size; ++i)
      if  (values[i] > 0)
          count_vector[values[i]]++;

  int md = 0;
  int freq = 0;

  for (int i = 0; i < NUM_CLASSES; i++)
      if (count_vector[i] > freq) {
          freq = count_vector[i];
          md = i;
      }
  return md;
}

// find mode of in an vector
__device__
float mode(const int *values, int size) {
  int count_vector[NUM_CLASSES] = {0};

  for (int i = 0; i < size; ++i)
          count_vector[values[i]]++;

  int md = 0;
  int freq = 0;

  for (int i = 0; i < NUM_CLASSES; i++)
      if (count_vector[i] > freq) {
          freq = count_vector[i];
          md = i;
      }
  return md;
}

__global__
void Downsample_Kernel( int *in_vox_size, int *out_vox_size,
                        int *in_labels, float *in_tsdf, float * in_grid_GPU,
                        int *out_labels, float *out_tsdf,
                        int label_downscale, float *out_grid_GPU) {

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;


    if (vox_idx >= out_vox_size[0] * out_vox_size[1] * out_vox_size[2]){
      return;
    }

    int down_size = label_downscale * label_downscale * label_downscale;

    //printf("down_size %d\n",down_size);

    int emptyT = int((0.95 * down_size)); //Empty Threshold

    int z = (vox_idx / ( out_vox_size[0] * out_vox_size[1]))%out_vox_size[2] ;
    int y = (vox_idx / out_vox_size[0]) % out_vox_size[1];
    int x = vox_idx % out_vox_size[0];

    //printf("x:%d, y:%d, z:%d\n", x, y, z);

    int label_vals[MAX_DOWN_SIZE] = {0};
    int count_vals=0;
    float tsdf_val = 0;

    int num_255 =0;

    int zero_count = 0;
    int zero_surface_count = 0;
    for (int tmp_x = x * label_downscale; tmp_x < (x + 1) * label_downscale; ++tmp_x) {
      for (int tmp_y = y * label_downscale; tmp_y < (y + 1) * label_downscale; ++tmp_y) {
        for (int tmp_z = z * label_downscale; tmp_z < (z + 1) * label_downscale; ++tmp_z) {
          int tmp_vox_idx = tmp_z * in_vox_size[0] * in_vox_size[1] + tmp_y * in_vox_size[0] + tmp_x;
          label_vals[count_vals] = int(in_labels[tmp_vox_idx]);
          count_vals += 1;

          if (in_labels[tmp_vox_idx] == 0 || in_labels[tmp_vox_idx] == 255) {
            if (in_labels[tmp_vox_idx]==255)
               num_255++;
            zero_count++;
          }
          if (in_grid_GPU[tmp_vox_idx] == 0 || in_labels[tmp_vox_idx] == 255) {
            zero_surface_count++;
          }

          tsdf_val += in_tsdf[tmp_vox_idx];

        }
      }
    }


    if (zero_count > emptyT) {
      out_labels[vox_idx] = float(mode(label_vals, down_size));
    } else {
      out_labels[vox_idx] = float(modeLargerZero(label_vals, down_size)); // object label mode without zeros
    }

    if (zero_surface_count > emptyT) {
      out_grid_GPU[vox_idx] = 0;
    } else {
      out_grid_GPU[vox_idx] = 1.0;
    }

    out_tsdf[vox_idx] = tsdf_val /  down_size;

    //Encode weights into downsampled labels


}



void DownsampleLabel_CPP(int *vox_size,
                         int out_scale,
                         int *segmentation_label_fullscale,
                         float *vox_tsdf_fullscale,
                         int *segmentation_label_downscale,
                         float *vox_weights,float *vox_vol, float *vox_grid) {

  //downsample lable
  clock_tick t1 = start_timer();

  int num_voxels_in = vox_size[0] * vox_size[1] * vox_size[2];
  int label_downscale = 4;
  int num_voxels_down = num_voxels_in/(label_downscale*label_downscale*label_downscale);
  int out_vox_size[3];

  float *vox_tsdf = new float[num_voxels_down];
  float *vox_grid_downscale = new float[num_voxels_down];

  out_vox_size[0] = vox_size[0]/label_downscale;
  out_vox_size[1] = vox_size[1]/label_downscale;
  out_vox_size[2] = vox_size[2]/label_downscale;

  int *in_vox_size_GPU;
  int *out_vox_size_GPU;
  int *in_labels_GPU;
  int *out_labels_GPU;
  float *in_tsdf_GPU;
  float *out_tsdf_GPU;
  float *in_grid_GPU;
  float *out_grid_GPU;

  cudaMalloc(&in_vox_size_GPU, 3 * sizeof(int));
  cudaMalloc(&out_vox_size_GPU, 3 * sizeof(int));
  cudaMalloc(&in_labels_GPU, num_voxels_in * sizeof(int));
  cudaMalloc(&in_tsdf_GPU, num_voxels_in * sizeof(float));
  cudaMalloc(&in_grid_GPU, num_voxels_in * sizeof(float));
  cudaMalloc(&out_labels_GPU, num_voxels_down * sizeof(int));
  cudaMalloc(&out_tsdf_GPU, num_voxels_down * sizeof(float));
  cudaMalloc(&out_grid_GPU, num_voxels_down * sizeof(float));

  cudaMemcpy(in_vox_size_GPU, vox_size,  3 * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(out_vox_size_GPU, out_vox_size,  3 * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(in_labels_GPU, segmentation_label_fullscale, num_voxels_in * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(in_tsdf_GPU, vox_tsdf_fullscale, num_voxels_in * sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpy(in_grid_GPU, vox_grid, num_voxels_in * sizeof(float), cudaMemcpyHostToDevice);


  int BLOCK_NUM = int((num_voxels_down + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  Downsample_Kernel<<< BLOCK_NUM, NUM_THREADS >>>(in_vox_size_GPU, out_vox_size_GPU,
                                                  in_labels_GPU, in_tsdf_GPU, in_grid_GPU,
                                                  out_labels_GPU, out_tsdf_GPU,
                                                  label_downscale, out_grid_GPU);

  cudaDeviceSynchronize();

  end_timer(t1,"Downsample duration");

  cudaMemcpy(segmentation_label_downscale, out_labels_GPU, num_voxels_down * sizeof(int), cudaMemcpyDeviceToHost);
  cudaMemcpy(vox_tsdf, out_tsdf_GPU, num_voxels_down * sizeof(float), cudaMemcpyDeviceToHost);
  cudaMemcpy(vox_grid_downscale, out_grid_GPU, num_voxels_down * sizeof(float), cudaMemcpyDeviceToHost);

  cudaFree(in_vox_size_GPU);
  cudaFree(out_vox_size_GPU);
  cudaFree(in_labels_GPU);
  cudaFree(out_labels_GPU);
  cudaFree(in_tsdf_GPU);
  cudaFree(out_tsdf_GPU);
  cudaFree(in_grid_GPU);
  cudaFree(out_grid_GPU);


  // Find number of occupied voxels
  // Save voxel indices of background
  // Set label weights of occupied voxels as 1
  int num_occ_voxels = 0; //Occupied voxels in occluded regions
  std::vector<int> bg_voxel_idx;

  memset(vox_weights, 0, num_voxels_down * sizeof(float));
  memset(vox_vol, 0, num_voxels_down * sizeof(float));

  for (int i = 0; i < num_voxels_down; ++i) {
      if ((segmentation_label_downscale[i]) > 0 && (segmentation_label_downscale[i]<255)) { //Occupied voxels in the room
          vox_weights[i] = 1.0;
          num_occ_voxels++;
      } else {
          if ((vox_tsdf[i] < 0) && (segmentation_label_downscale[i]<255)) {
              bg_voxel_idx.push_back(i); // background voxels in unobserved region in the room
          }
      }

      if ((vox_grid_downscale[i] > 0) && (segmentation_label_downscale[i]>0) && (segmentation_label_downscale[i]<255)) { //Occupied voxels in the room
          vox_vol[i] = 0.5;
      } else {
          if ((vox_tsdf[i] < 0.1) && (segmentation_label_downscale[i]<255)) {
              if ((vox_tsdf[i] > -0.7) && (segmentation_label_downscale[i]>0))
                 vox_vol[i] = -0.5;
              else
                 vox_vol[i] = -1;
          } else {
                 vox_vol[i] = 1;
          }

      }

      if (vox_vol[i] == 0)
             vox_vol[i] = -3;
      if (vox_tsdf[i] > 1) {
             vox_weights[i] = 0;
             vox_vol[i] = -2;
      }
      if (segmentation_label_downscale[i] == 255){  //outside room
          segmentation_label_downscale[i] = 0;
          vox_vol[i] = -4;
      }


  }

  float occluded_empty_weight = num_occ_voxels * sample_neg_obj_ratio / bg_voxel_idx.size();

  for (int i = 0; i < bg_voxel_idx.size(); ++i) {
     vox_weights[bg_voxel_idx[i]] = occluded_empty_weight;
  }

  end_timer(t1,"Downsample duration + copy");

  delete [] vox_tsdf;


}

__global__
void depth2Grid_edges(float *cam_pose, int *vox_size,  float *vox_origin, float *depth_data, unsigned char *edges_data,
                      float *vox_edges, float *parameters_GPU){


  float *cam_K_GPU;
  int frame_width_GPU, frame_height_GPU;
  float vox_unit_GPU, vox_margin_GPU;

  get_parameters_GPU(parameters_GPU, &cam_K_GPU, &frame_width_GPU, &frame_height_GPU,
                                     &vox_unit_GPU, &vox_margin_GPU);


  // Get point in world coordinate
  // Try to parallel later

  // Get point in world coordinate
  int pixel_x = blockIdx.x;
  int pixel_y = threadIdx.x;



  unsigned char point_edges = edges_data[pixel_y * frame_width_GPU + pixel_x];

  if (point_edges > 0) {

      float min_depth = depth_data[pixel_y * frame_width_GPU + pixel_x];
      int min_x = pixel_x;
      int min_y = pixel_y;

      //Search for the closest depth around the edge to get the object at the foreground
      for (int x =  pixel_x - 1; x<=pixel_x+1; x++) {
          if (x>=0 & x<frame_width_GPU) {
              for (int y = pixel_y -1; y<=pixel_y+1; y++) {
                   if (y>=0 & y<frame_height_GPU) {

                          float point_depth = depth_data[y * frame_width_GPU + x];
                          if (point_depth < min_depth) {
                                   min_depth = point_depth;
                                   min_x = x;
                                   min_y = y;
                          }
                   }
              }
          }

      }


      float point_cam[3] = {0};
      point_cam[0] =  (min_x - cam_K_GPU[2])*min_depth/cam_K_GPU[0];
      point_cam[1] =  (min_y - cam_K_GPU[5])*min_depth/cam_K_GPU[4];
      point_cam[2] =  min_depth;

      float point_base[3] = {0};

      point_base[0] = cam_pose[0 * 4 + 0]* point_cam[0] + cam_pose[0 * 4 + 1]*  point_cam[1] + cam_pose[0 * 4 + 2]* point_cam[2];
      point_base[1] = cam_pose[1 * 4 + 0]* point_cam[0] + cam_pose[1 * 4 + 1]*  point_cam[1] + cam_pose[1 * 4 + 2]* point_cam[2];
      point_base[2] = cam_pose[2 * 4 + 0]* point_cam[0] + cam_pose[2 * 4 + 1]*  point_cam[1] + cam_pose[2 * 4 + 2]* point_cam[2];

      point_base[0] = point_base[0] + cam_pose[0 * 4 + 3];
      point_base[1] = point_base[1] + cam_pose[1 * 4 + 3];
      point_base[2] = point_base[2] + cam_pose[2 * 4 + 3];


      //printf("vox_origin: %f,%f,%f\n",vox_origin[0],vox_origin[1],vox_origin[2]);
      // World coordinate to grid coordinate
      int z = (int)floor((point_base[0] - vox_origin[0])/ vox_unit_GPU);
      int x = (int)floor((point_base[1] - vox_origin[1])/ vox_unit_GPU);
      int y = (int)floor((point_base[2] - vox_origin[2])/ vox_unit_GPU);
      //printf("point_base: %f,%f,%f, %d,%d,%d, %d,%d,%d \n",point_base[0],point_base[1],point_base[2], z, x, y, vox_size[0],vox_size[1],vox_size[2]);

      // mark vox_out with 1.0
      if( x >= 0 && x < vox_size[0] && y >= 0 && y < vox_size[1] && z >= 0 && z < vox_size[2]){
          int vox_idx = z * vox_size[0] * vox_size[1] + y * vox_size[0] + x;
          vox_edges[vox_idx] = float(1.0);
      }
  }
}







__global__
void SquaredDistanceTransform(float *cam_pose, int *vox_size,  float *vox_origin, float *depth_data, float *vox_grid,
                              float *vox_tsdf, float *parameters_GPU) {

    float *cam_K_GPU = parameters_GPU;
    int frame_width_GPU= int(parameters_GPU[9]), frame_height_GPU= int(parameters_GPU[10]);
    float vox_unit_GPU= parameters_GPU[11], vox_margin_GPU = parameters_GPU[12];

    int search_region = (int)round(vox_margin_GPU/vox_unit_GPU);

    int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (vox_idx >= vox_size[0] * vox_size[1] * vox_size[2]){
      return;
    }

    if (vox_grid[vox_idx] >0 ){
       vox_tsdf[vox_idx] = 0;
       return;
    }

    int z = (vox_idx / ( vox_size[0] * vox_size[1]))%vox_size[2] ;
    int y = (vox_idx / vox_size[0]) % vox_size[1];
    int x = vox_idx % vox_size[0];

    // Get point in world coordinates XYZ -> YZX
    float point_base[3] = {0};
    point_base[0] = float(z) * vox_unit_GPU + vox_origin[0];
    point_base[1] = float(x) * vox_unit_GPU + vox_origin[1];
    point_base[2] = float(y) * vox_unit_GPU + vox_origin[2];

    // Encode height from floor ??? check later

    // Get point in current camera coordinates
    float point_cam[3] = {0};
    point_base[0] = point_base[0] - cam_pose[0 * 4 + 3];
    point_base[1] = point_base[1] - cam_pose[1 * 4 + 3];
    point_base[2] = point_base[2] - cam_pose[2 * 4 + 3];
    point_cam[0] = cam_pose[0 * 4 + 0] * point_base[0] + cam_pose[1 * 4 + 0] * point_base[1] + cam_pose[2 * 4 + 0] * point_base[2];
    point_cam[1] = cam_pose[0 * 4 + 1] * point_base[0] + cam_pose[1 * 4 + 1] * point_base[1] + cam_pose[2 * 4 + 1] * point_base[2];
    point_cam[2] = cam_pose[0 * 4 + 2] * point_base[0] + cam_pose[1 * 4 + 2] * point_base[1] + cam_pose[2 * 4 + 2] * point_base[2];
    if (point_cam[2] <= 0)
      return;

    // Project point to 2D
    int pixel_x = roundf(cam_K_GPU[0] * (point_cam[0] / point_cam[2]) + cam_K_GPU[2]);
    int pixel_y = roundf(cam_K_GPU[4] * (point_cam[1] / point_cam[2]) + cam_K_GPU[5]);
    if (pixel_x < 0 || pixel_x >= frame_width_GPU || pixel_y < 0 || pixel_y >= frame_height_GPU){ // outside FOV
      //vox_tsdf[vox_idx] = GPUCompute2StorageT(-1.0);
      vox_tsdf[vox_idx] = 2000;
      return;
    }

    // Get depth
    float point_depth = depth_data[pixel_y * frame_width_GPU + pixel_x];
    if (point_depth < float(0.5f) || point_depth > float(8.0f))
    {
      vox_tsdf[vox_idx] = 1;
      return;
    }
    if (roundf(point_depth) == 0){ // mising depth
      vox_tsdf[vox_idx] = -1.0;
      return;
    }

    // Get depth difference
    float point_dist = (point_depth - point_cam[2]) * sqrtf(1 + powf((point_cam[0] / point_cam[2]), 2) + powf((point_cam[1] / point_cam[2]), 2));
    //float sign = point_dist/abs(point_dist);

    float sign;
    if (abs(point_depth - point_cam[2]) < 0.0001){
        sign = 1; // avoid NaN
    }else{
        sign = (point_depth - point_cam[2])/abs(point_depth - point_cam[2]);
    }
    vox_tsdf[vox_idx] = sign;

    int radius=search_region; // out -> in
    int found = 0;
    //fixed y planes
    int iiy = max(0,y-radius);
    for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iiy = min(y+radius,vox_size[1]);
    for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed x planes
    int iix = max(0,x-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iix = min(x+radius,vox_size[0]);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
        for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    //fixed z planes
    int iiz = max(0,z-radius);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }
    iiz = min(z+radius,vox_size[2]);
    for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
        for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
            int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
            if (vox_grid[iidx] > 0){
              found = 1;
              float xd = abs(x - iix);
              float yd = abs(y - iiy);
              float zd = abs(z - iiz);
              float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
              if (tsdf_value < abs(vox_tsdf[vox_idx])){
                vox_tsdf[vox_idx] = tsdf_value*sign;
              }
            }
        }
    }


    if (found == 0)
        return;

    radius=1; // in -> out
    found = 0;
    while (radius < search_region) {
        //fixed y planes
        int iiy = max(0,y-radius);
        for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiy = min(y+radius,vox_size[1]);
        for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed x planes
        int iix = max(0,x-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iix = min(x+radius,vox_size[0]);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
            for (int iiz = max(0,z-radius); iiz < min((int)vox_size[2],z+radius+1); iiz++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        //fixed z planes
        int iiz = max(0,z-radius);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        iiz = min(z+radius,vox_size[2]);
        for (int iiy = max(0,y-radius); iiy < min((int)vox_size[1],y+radius+1); iiy++){
            for (int iix = max(0,x-radius); iix < min((int)vox_size[0],x+radius+1); iix++){
                int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
                if (vox_grid[iidx] > 0){
                  found = 1;
                  float xd = abs(x - iix);
                  float yd = abs(y - iiy);
                  float zd = abs(z - iiz);
                  float tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/search_region;
                  if (tsdf_value < abs(vox_tsdf[vox_idx])){
                    vox_tsdf[vox_idx] = tsdf_value*sign;
                  }
                }
            }
        }
        if (found == 1)
          return;

        radius++;

    }
}



void ComputeTSDF_edges_CPP(int *vox_size,  unsigned char *depth_image, unsigned char *edges_image,
                     float *vox_grid, float *vox_tsdf, float *vox_edges, float *tsdf_edges) {

  //cout << "\nComputeTSDF_CPP\n";
  clock_tick t1 = start_timer();


  int num_voxels = vox_size[0] * vox_size[1] * vox_size[2];

  float *depth_data_GPU, *vox_grid_GPU, *vox_tsdf_GPU, *vox_edges_GPU, *tsdf_edges_GPU;
  unsigned char *edges_data_GPU;
  int *vox_size_GPU;

  cudaMalloc(&vox_size_GPU, 3 * sizeof(int));

  cudaMalloc(&depth_data_GPU, frame_height * frame_width * sizeof(float));
  //cudaMalloc(&edges_data_GPU, frame_height * frame_width * sizeof(float));
  cudaMalloc(&vox_grid_GPU, num_voxels * sizeof(float));
  //cudaMalloc(&vox_tsdf_GPU, num_voxels * sizeof(float));
  //cudaMalloc(&vox_edges_GPU, num_voxels * sizeof(float));
  //cudaMalloc(&tsdf_edges_GPU, num_voxels * sizeof(float));
  //cudaMemset(vox_tsdf_GPU, 0, num_voxels * sizeof(float));
  //cudaMemset(tsdf_edges_GPU, 0, num_voxels * sizeof(float));
  //cudaMemset(vox_edges_GPU, 0, num_voxels * sizeof(float));

  cudaMemcpy(vox_size_GPU, vox_size, 3 * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(depth_data_GPU, depth_image, frame_height * frame_width * sizeof(float), cudaMemcpyHostToDevice);
  //cudaMemcpy(edges_data_GPU, edges_image, frame_height * frame_width * 1, cudaMemcpyHostToDevice);


  end_timer(t1, "Prepare duration");


  t1 = start_timer();
  // from depth map to binaray voxel representation
  depth2Grid<<<frame_width,frame_height>>>(vox_size_GPU,  depth_data_GPU,
                                           vox_grid_GPU, parameters_GPU);
  cudaDeviceSynchronize();


  depth2Grid_edges<<<frame_width,frame_height>>>(cam_pose_GPU, vox_size_GPU,  vox_origin_GPU, depth_data_GPU, edges_data_GPU,
                                           vox_edges_GPU, parameters_GPU);
  cudaDeviceSynchronize();
  end_timer(t1,"depth2Grid duration");

  // distance transform
  int BLOCK_NUM = int((num_voxels + size_t(NUM_THREADS) - 1) / NUM_THREADS);

  t1 = start_timer();

  SquaredDistanceTransform<<< BLOCK_NUM, NUM_THREADS >>>(cam_pose_GPU, vox_size_GPU,  vox_origin_GPU, depth_data_GPU, vox_grid_GPU, vox_tsdf_GPU, parameters_GPU);
  cudaDeviceSynchronize();

  SquaredDistanceTransform<<< BLOCK_NUM, NUM_THREADS >>>(cam_pose_GPU, vox_size_GPU,  vox_origin_GPU, depth_data_GPU, vox_edges_GPU, tsdf_edges_GPU, parameters_GPU);
  cudaDeviceSynchronize();

  end_timer(t1,"SquaredDistanceTransform duration");

  t1 = start_timer();
  cudaMemcpy(vox_grid, vox_grid_GPU, num_voxels * sizeof(float), cudaMemcpyDeviceToHost);
  cudaMemcpy(vox_edges, vox_edges_GPU, num_voxels * sizeof(float), cudaMemcpyDeviceToHost);
  cudaMemcpy(vox_tsdf, vox_tsdf_GPU, num_voxels * sizeof(float), cudaMemcpyDeviceToHost);
  cudaMemcpy(tsdf_edges, tsdf_edges_GPU, num_voxels * sizeof(float), cudaMemcpyDeviceToHost);




  cudaFree(vox_size_GPU);
  cudaFree(depth_data_GPU);

  //cudaFree(edges_data_GPU);
  cudaFree(vox_grid_GPU);
  //cudaFree(vox_edges_GPU);
  //cudaFree(vox_tsdf_GPU);
  //cudaFree(tsdf_edges_GPU);

  end_timer(t1,"closeup duration");

}

void FlipTSDF_CPP( int *vox_size, float *vox_tsdf){

  clock_tick t1 = start_timer();

  for (int vox_idx=0; vox_idx< vox_size[0]*vox_size[1]*vox_size[2]; vox_idx++) {

      float value = float(vox_tsdf[vox_idx]);
      if (value > 1)
          value =1;


      float sign;
      if (abs(value) < 0.001)
        sign = 1;
      else
        sign = value/abs(value);

      vox_tsdf[vox_idx] = sign*(max(0.001,(1.0-abs(value))));
  }
  end_timer(t1,"FlipTSDF");
}

void ProcessEdges_CPP(int *vox_size,
                 int out_scale,
                 unsigned char *depth_data,
                 unsigned char *edges_data,
                 float *vox_tsdf,
                 float *vox_edges,
                 float *tsdf_edges,
                 float *vox_limits,
                 int *segmentation_label_downscale) {


    int num_voxels = vox_size[0] * vox_size[1] * vox_size[2];

    int *segmentation_label_fullscale;
    segmentation_label_fullscale= (int *) malloc((vox_size[0]*vox_size[1]*vox_size[2]) * sizeof(int));

    float *vox_grid = new float[num_voxels];
    memset(vox_grid, 0, num_voxels * sizeof(float));

    ComputeTSDF_edges_CPP(vox_size,  depth_data, edges_data, vox_grid, vox_tsdf, vox_edges, tsdf_edges);

  DownsampleLabel_CPP(vox_size,
                            out_scale,
                            segmentation_label_fullscale,
                            vox_tsdf,
                            segmentation_label_downscale,
                            vox_weights,vox_vol,vox_grid);


    FlipTSDF_CPP( vox_size, vox_tsdf);
    FlipTSDF_CPP( vox_size, tsdf_edges);

    delete [] vox_grid;

    free(segmentation_label_fullscale);
    //FlipTSDF_CPP( out_vox_size, vox_vol);


}
*/


